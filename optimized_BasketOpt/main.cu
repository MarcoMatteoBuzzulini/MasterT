#include "hip/hip_runtime.h"
//
//  MonteCarlo.cu
//  tesi
//
//  Created by Marco Matteo Buzzulini on 27/11/17.
//  Copyright © 2017 Marco Matteo Buzzulini. All rights reserved.
//

#include "MonteCarlo.h"
#include <hip/hip_runtime.h>

// includes, project
#include <helper_functions.h> // Helper functions (utilities, parsing, timing)
#include <hip/hip_runtime_api.h>      // helper functions (cuda error checking and initialization)
#include <multithreading.h>

//	Host utility functions declarations
void Chol( double c[N][N], double a[N][N] );

//	Host MonteCarlo
extern "C" OptionValue host_basketOpt(MultiOptionData*, int);

//	Device MonteCarlo
extern "C" OptionValue dev_basketOpt(MultiOptionData *, int, int);

///////////////////////////////////
//	PRINT FUNCTIONS
///////////////////////////////////
void printVect( double *mat, int c ){
    int i,j,r=1;
    for(i=0; i<r; i++){
        printf("\n!\t");
        for(j=0; j<c; j++){
            printf("\t%f\t",mat[j+i*c]);
        }
        printf("\t!");
    }
    printf("\n\n");
}

void printOption( OptionData o){
    printf("\n-\tOption data\t-\n\n");
    printf("Underlying asset price:\t € %.2f\n", o.s);
    printf("Strike price:\t\t € %.2f\n", o.k);
    printf("Risk free interest rate: %.2f %%\n", o.r * 100);
    printf("Volatility:\t\t\t %.2f %%\n", o.v * 100);
    printf("Time to maturity:\t\t %.2f %s\n", o.t, (o.t>1)?("years"):("year"));
}

void printMat( double *mat, int r, int c ){
    int i,j;
    for(i=0; i<r; i++){
        printf("\n!\t");
        for(j=0; j<c; j++){
            printf("\t%f\t",mat[j+i*c]);
        }
        printf("\t!");
    }
    printf("\n\n");
}

void printMultiOpt( MultiOptionData *o){
    printf("\n-\tBasket Option data\t-\n\n");
    printf("Number of assets: %d\n",N);
    printf("Underlying assets prices:\n");
    printVect(o->s, N);
    printf("Volatility:\n");
    printVect(o->v, N);
    printf("Weights:");
    printVect(o->w, N);
    printf("Correlation matrix:\n");
    printMat(&o->p[0][0], N, N);
    printf("Strike price:\t € %.2f\n", o->k);
    printf("Risk free interest rate: %.2f \n", o->r);
    printf("Time to maturity:\t %.2f %s\n", o->t, (o->t>1)?("years"):("year"));
}

///////////////////////////////////
//	ADJUST FUNCTIONS
///////////////////////////////////

void sizeAdjust(hipDeviceProp_t *deviceProp, int *numBlocks, int *numThreads){
	int maxGridSize = deviceProp->maxGridSize[0];
	int maxBlockSize = deviceProp->maxThreadsPerBlock;
	//	Replacing in case of wrong size
	if(*numBlocks > maxGridSize){
		*numBlocks = maxGridSize;
		printf("Warning: maximum size of Grid is %d",*numBlocks);
	}
	if(*numThreads > maxBlockSize){
		*numThreads = maxBlockSize;
		printf("Warning: maximum size of Blocks is %d",*numThreads);
	}
}

void memAdjust(hipDeviceProp_t *deviceProp, int *numThreads){
		size_t maxShared = deviceProp->sharedMemPerBlock;
		size_t maxConstant = deviceProp->totalConstMem;
		int sizeDouble = sizeof(double);
		int numShared = sizeDouble * *numThreads * 2;
		if(sizeof(MultiOptionData) > maxConstant){
			printf("\nWarning: Excess use of constant memory: %zu\n",maxConstant);
			printf("A double variable size is: %d\n",sizeDouble);
			printf("In a MultiOptionData struct there's a consumption of %zu constant memory\n",sizeof(MultiOptionData));
			printf("In this Basket Option there's %d stocks\n",N);
			int maxDim = (int)maxConstant/(sizeDouble*8);
			printf("The optimal number of dims should be: %d stocks\n",maxDim);
		}
		if(numShared > maxShared){
			printf("\nWarning: Excess use of shared memory: %zu\n",maxShared);
			printf("A double variable size is: %d\n",sizeDouble);
			int maxThreads = (int)maxShared / (2*sizeDouble);
			printf("The optimal number of thread should be: %d\n",maxThreads);
		}
		printf("\n");
}

void optimalAdjust(hipDeviceProp_t *deviceProp, int *numBlocks, int *numThreads){
	int multiProcessors = deviceProp->multiProcessorCount;
	int cudaCoresPM = _ConvertSMVer2Cores(deviceProp->major, deviceProp->minor);
	*numBlocks = multiProcessors * 40;
	*numThreads = pow(2,(int)(log(cudaCoresPM)/log(2)));
	sizeAdjust(deviceProp,numBlocks, numThreads);
}

void choseParameters(int *numBlocks, int *numThreads){
		hipDeviceProp_t deviceProp;
		CudaCheck(hipGetDeviceProperties(&deviceProp, 0));
		char risp;
		printf("\nParametri CUDA:\n");
		printf("Scegli il numero di Blocchi: ");
		scanf("%d",numBlocks);
		printf("Scegli il numero di Threads per blocco: ");
		scanf("%d",numThreads);
		printf("Vuoi ottimizzare i parametri inseriti? (Y/N) ");
		scanf("%s",&risp);
		if(risp=='Y')
			optimalAdjust(&deviceProp,numBlocks, numThreads);
		else
			sizeAdjust(&deviceProp,numBlocks, numThreads);
		memAdjust(&deviceProp,numThreads);
}

////////////////////////////////////////////////////////////////////////////////////////
//                                      MAIN
////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, const char * argv[]) {
    /*--------------------------- DATA INSTRUCTION -----------------------------------*/
	double dw = (double)1/(double)N;
	MultiOptionData option;
	//	Volatility
	option.v[0] = 0.2;
	option.v[1] = 0.3;
	option.v[2] = 0.2;
	//	Spot prices
	option.s[0] = 100;
	option.s[1] = 100;
	option.s[2] = 100;
	//	Weights
	option.w[0] = dw;
	option.w[1] = dw;
	option.w[2] = dw;
	//	Correlations
	option.p[0][0] = 1;
			option.p[0][1] = -0.5;
					option.p[0][2] = -0.5;
	option.p[1][0] = -0.5;
			option.p[1][1] = 1;
					option.p[1][2] = -0.5;
	option.p[2][0] = -0.5;
			option.p[2][1] = -0.5;
					option.p[2][2] = 1;
	//	Drift vectors for the brownians
	option.d[0] = 0;
	option.d[1] = 0;
	option.d[2] = 0;
	option.k= 100.f;
	option.r= 0.048790164;
	option.t= 1.f;
	printf("Basket Option Pricing\n");

	//	Definizione dei parametri CUDA per l'esecuzione in parallelo
	int numBlocks, numThreads;
	choseParameters(&numBlocks, &numThreads);

	printf("Simulazione di ( %d ; %d )\n",numBlocks, numThreads);
	int SIMS = numBlocks*PATH;

	//	Print Option details
	printMultiOpt(&option);

    /*---------------- CORE COMPUTATIONS ----------------*/
    //	Cholevski factorization
    double cholRho[N][N];
    int i,j;
    Chol(option.p, cholRho);
    for(i=0;i<N;i++)
    	for(j=0;j<N;j++)
           	option.p[i][j]=cholRho[i][j];
    OptionValue CPU_sim, GPU_sim;
    
    float CPU_timeSpent=0, GPU_timeSpent=0, speedup;
    double price;
    //clock_t h_start, h_stop;
    hipEvent_t d_start, d_stop;
    CudaCheck( hipEventCreate( &d_start ));
    CudaCheck( hipEventCreate( &d_stop ));

    /* CPU Monte Carlo
    printf("\nMonte Carlo execution on CPU:\nN^ simulations: %d\n\n",SIMS);
    h_start = clock();
    //CudaCheck( hipEventRecord( d_start, 0 ));
    CPU_sim=host_basketOpt(&option, SIMS);
    h_stop = clock();
    CPU_timeSpent = ((float)(h_stop - h_start)) / CLOCKS_PER_SEC;
    //CudaCheck( hipEventRecord( d_stop, 0));
    //CudaCheck( hipEventSynchronize( d_stop ));
    //CudaCheck( hipEventElapsedTime( &CPU_timeSpent, d_start, d_stop ));
    //CPU_timeSpent /= CLOCKS_PER_SEC;
    
    price = CPU_sim.Expected;
    printf("Simulated price for the basket option: € %f with I.C [ %f;%f ]\n", price, price - CPU_sim.Confidence, price + CPU_sim.Confidence);
    printf("Total execution time: %f s\n\n", CPU_timeSpent);
    */
    // GPU Monte Carlo
    printf("\nMonte Carlo execution on GPU:\nN^ simulations: %d\n",SIMS);
    CudaCheck( hipEventRecord( d_start, 0 ));
    GPU_sim = dev_basketOpt(&option, numBlocks, numThreads);
    CudaCheck( hipEventRecord( d_stop, 0));
    CudaCheck( hipEventSynchronize( d_stop ));
    CudaCheck( hipEventElapsedTime( &GPU_timeSpent, d_start, d_stop ));
    GPU_timeSpent /= 1000;
    
    price = GPU_sim.Expected;
    printf("Simulated price for the basket option: € %f with I.C [ %f;%f ]\n", price, price-GPU_sim.Confidence, price + GPU_sim.Confidence);
    printf("Total execution time: %f s\n\n", GPU_timeSpent);
    
    // Comparing time spent with the two methods
    printf( "-\tComparing results:\t-\n");
    speedup = abs(CPU_timeSpent / GPU_timeSpent);
    printf( "The GPU's speedup: %.2f \n", speedup);
    return 0;
}

void Chol( double c[N][N], double a[N][N] ){
    int i,j,k;
    double v[N];
    for( i=0; i<N; i++){
        for( j=0; j<N; j++ ){
            if( j>=i ){
                //Triangolare inferiore
            	//v[j]=c[j][i]
            	v[j]=c[j][i];
                for(k=0; k<i; k++)    //Scorre tutta
                    //v[j] = v[j] - a[i][k] * a[j][k]
                    v[j] = v[j] - a[i][k] * a[j][k];
                //a[j][i] = v[j] / sqrt( v[i] )
                if(v[i]>0)
                	a[j][i] = v[j] / sqrt( v[i] );
                else
                	a[j][i] = 0.0f;
            }
            else
                //Triangolare superiore a[j][i]
            	a[j][i] = 0.0f;
        }
    }
}
