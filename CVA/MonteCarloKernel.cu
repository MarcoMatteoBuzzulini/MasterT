#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

//#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "MonteCarlo.h"

__device__ __constant__ double D_DRIFTVECT[N], D_CHOLMAT[N][N], D_S[N], D_V[N], D_W[N], D_K, D_T, D_R;

__device__ void prodConstMat(Matrix *second, Matrix *result){
    if(N != second->rows){
        printf("Non si può effettuare la moltiplicazione\n");
        return;
    }
    double somma;
    int i,j,k;
    result->rows = N;
    result->cols = second->cols;
    for(i=0;i<result->rows;i++){
        for(j=0;j<result->cols;j++){
            somma = 0;
            for(k=0;k<N;k++)
                //somma += first->data[i][k]*second->data[k][j];
                somma += D_CHOLMAT[i][k] * second->data[j+k*second->cols];
            //result->data[i][j] = somma;
            result->data[j+i*result->cols] = somma;
        }
    }
}

__device__ void devGaussVect(hiprandState *threadState, double *result, const int n){
    int i;
    // Random number vector
    double g[N];
    // RNGs
    for(i=0;i<n;i++)
        g[i]=hiprand_normal(threadState);
    Matrix gauss, r;
    gauss.rows = n;     r.rows=n;
    gauss.cols = 1;     r.cols=1;
    gauss.data = &g[0]; r.data=result;
    //A*G
    prodConstMat(&gauss,&r);
    //X=m+A*G
    for(i=0;i<n;i++){
        r.data[i] += D_DRIFTVECT[i];
    }
}

__device__ void devMultiStVal(double *values, double *g, double t, double r, int n){
    int i;
    for(i=0;i<n;i++){
        double mu = (r - 0.5 * D_V[i] * D_V[i])*t;
        double si = D_V[i] * g[i] * sqrt(t);
        values[i] = D_S[i] * exp(mu+si);
    }
}

__global__ void MultiMCBasketOptKernel(hiprandState * randseed, OptionValue *d_CallValue){
    int i,j;
    int cacheIndex = threadIdx.x;
    int blockIndex = blockIdx.x;
    /*------------------ SHARED MEMORY DICH ----------------*/
    __shared__ double s_Sum[MAX_THREADS];
    __shared__ double s_Sum2[MAX_THREADS];

    //Monte Carlo variables
    double st_sum=0.0f, price;

    //vectors of brownian and ST
    double bt[N];
    double s[N];

    // Global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Copy random number state to local memory
    hiprandState threadState = randseed[tid];

    OptionValue sum = {0, 0};

    for( i=cacheIndex; i<SIM; i+=blockDim.x){
        st_sum = 0;
        //Simulation of stock prices
        devGaussVect(&threadState,bt,N);
        devMultiStVal(s, bt, D_T, D_R, N);
        for(j=0;j<N;j++)
            st_sum += s[j] * D_W[j];
        //Option payoff
        price = st_sum - D_K;
        if(price<0)
            price = 0.0f;
        sum.Expected += price;
        sum.Confidence += price*price;
    }
    s_Sum[cacheIndex] = sum.Expected;
    s_Sum2[cacheIndex] = sum.Confidence;
    __syncthreads();
    //Reduce shared memory accumulators and write final result to global memory
    int halfblock = blockDim.x/2;
    do{
        if ( cacheIndex < halfblock ){
            s_Sum[cacheIndex] += s_Sum[cacheIndex+halfblock];
            s_Sum2[cacheIndex] += s_Sum2[cacheIndex+halfblock];
            __syncthreads();
        }
        halfblock /= 2;
    }while ( halfblock != 0 );
    __syncthreads();
    //Keeping the first element for each block using one thread
    if (threadIdx.x == 0){
    	d_CallValue[blockIndex].Expected = s_Sum[0];
    	d_CallValue[blockIndex].Confidence = s_Sum2[0];
    }
}

__global__ void randomSetup( hiprandState *randSeed ){
    // Global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Each threadblock gets different seed, threads within a threadblock get different sequence numbers
    hiprand_init(blockIdx.x + gridDim.x, threadIdx.x, 0, &randSeed[tid]);
}

extern "C" void GPUBasketOpt(MultiOptionData *option, OptionValue *callValue ){
    int i;
    /*----------------- HOST MEMORY -------------------*/
    OptionValue *h_CallValue;
    //Allocation pinned host memory for prices
    CudaCheck(hipHostAlloc(&h_CallValue, sizeof(OptionValue)*(MAX_BLOCKS),hipHostMallocDefault));

    /*--------------- CONSTANT MEMORY ----------------*/

    CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(D_DRIFTVECT),option->d,N*sizeof(double)));
    CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(D_CHOLMAT),option->p,N*N*sizeof(double)));
    CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(D_S),option->s,N*sizeof(double)));
    CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(D_V),option->v,N*sizeof(double)));
    CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(D_W),option->w,N*sizeof(double)));
    CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(D_K),&option->k,sizeof(double)));
    CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(D_T),&option->t,sizeof(double)));
    CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(D_R),&option->r,sizeof(double)));

    /*----------------- DEVICE MEMORY -------------------*/
    OptionValue *d_CallValue;
    CudaCheck(hipMalloc(&d_CallValue, sizeof(OptionValue)*(MAX_BLOCKS)));

    /*------------ RNGs and TIME VARIABLES --------------*/
    hiprandState *RNG;
    hipEvent_t start, stop;
    CudaCheck( hipEventCreate( &start ));
    CudaCheck( hipEventCreate( &stop ));
    float time;

    // RANDOM NUMBER GENERATION KERNEL
    //Allocate states for pseudo random number generators
    CudaCheck(hipMalloc((void **) &RNG, MAX_BLOCKS * MAX_THREADS * sizeof(hiprandState)));
    //Setup for the random number sequence
    randomSetup<<<MAX_BLOCKS, MAX_THREADS>>>(RNG);

    //MONTE CARLO KERNEL
    CudaCheck( hipEventRecord( start, 0 ));
    MultiMCBasketOptKernel<<<MAX_BLOCKS, MAX_THREADS>>>(RNG,(OptionValue *)(d_CallValue));
    CudaCheck( hipEventRecord( stop, 0));
    CudaCheck( hipEventSynchronize( stop ));
    CudaCheck( hipEventElapsedTime( &time, start, stop ));
    printf( "\nMonte Carlo simulations done in %f milliseconds\n", time);
    CudaCheck( hipEventDestroy( start ));
    CudaCheck( hipEventDestroy( stop ));

    //MEMORY CPY: prices per block
    CudaCheck(hipMemcpy(h_CallValue, d_CallValue, MAX_BLOCKS * sizeof(OptionValue), hipMemcpyDeviceToHost));

    // Closing Monte Carlo
    long double sum=0, sum2=0, price, empstd;
    long int nSim = MAX_BLOCKS * SIM;
    for ( i = 0; i < MAX_BLOCKS; i++ ){
        sum += h_CallValue[i].Expected;
        sum2 += h_CallValue[i].Confidence;
    }
    price = exp(-(option->r*option->t)) * (sum/(double)nSim);
    empstd = sqrt((double)((double)nSim * sum2 - sum * sum)
                         /((double)nSim * (double)(nSim - 1)));
    callValue->Confidence = 1.96 * empstd / (double)sqrt((double)nSim);
    callValue->Expected = price;

    //Free memory space
    CudaCheck(hipFree(RNG));
    CudaCheck(hipHostFree(h_CallValue));
    CudaCheck(hipFree(d_CallValue));
}
