#include "hip/hip_runtime.h"
//
//  MonteCarlo.cu
//  tesi
//
//  Created by Marco Matteo Buzzulini on 27/11/17.
//  Copyright © 2017 Marco Matteo Buzzulini. All rights reserved.
//

#include "MonteCarlo.h"


#define N 3
#define MAX_BLOCKS 1000
#define MAX_THREADS 1024
#define PATH 10000

//	Host utility functions declarations
double* Chol( double *c, int n);

///////////////////////////////////
//	PRINT FUNCTIONS
///////////////////////////////////
void printOption( OptionData o){
    printf("\n-\tOption data\t-\n\n");
    printf("Underlying asset price:\t € %.2f\n", o.s);
    printf("Strike price:\t\t € %.2f\n", o.k);
    printf("Risk free interest rate: %.2f %%\n", o.r * 100);
    printf("Volatility:\t\t\t %.2f %%\n", o.v * 100);
    printf("Time to maturity:\t\t %.2f %s\n", o.t, (o.t>1)?("years"):("year"));
}

void printMultiOpt( MultiOptionData *o){
    int n=o->n;
    printf("\n-\tBasket Option data\t-\n\n");
    printf("Number of assets: %d\n",n);
    printf("Underlying assets prices:\n");
    printVect(o->s, n);
    printf("Volatility:\n");
    printVect(o->v, n);
    printf("Weights:");
    printVect(o->w, n);
    printf("Correlation matrix:\n");
    printMat(o->p, n, n);
    printf("Strike price:\t € %.2f\n", o->k);
    printf("Risk free interest rate %.2f \n", o->r);
    printf("Time to maturity:\t %.2f %s\n", o->t, (o->t>1)?("years"):("year"));
}

////////////////////////////////////////////////////////////////////////////////////////
//                                      MAIN
////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, const char * argv[]) {
    /*--------------------------- DATA INSTRUCTION -----------------------------------*/
	const double
	    v[N]={ 0.2, 0.3, 0.2 },
	    s[N]={ 100, 100, 100 },
	    w[N]={ dw, dw, dw },
	    p[N][N]={
	        {   1,      -0.5,   -0.5  },
	        {   -0.5,   1,      -0.5  },
	        {   -0.5,    -0.5,    1   }
	    },
	    d[N]={0,0,0};
		K = 100.f;
		R = 0.048790164;
		T = 1.f;
		dw = (double)1/(double)N;
        
    /*--------------------------- CPU PATHULATION -----------------------------------*/
    int SIMS = MAX_BLOCKS*PATH;
    
    /*--------------------------------- MAIN ---------------------------------------*/
    MultiOptionData option;
    OptionValue CPU_sim, GPU_sim;
    
    float CPU_timeSpent, GPU_timeSpent, speedup;
    double price;
    clock_t h_start, h_stop;
    hipEvent_t d_start, d_stop;
    CudaCheck( hipEventCreate( &d_start ));
    CudaCheck( hipEventCreate( &d_stop ));
    
    //	Setting up the option
    option.s = s;
    option.v = v;
    option.p = &p[0][0];
    option.d = d;
    option.w = w;
    option.k = K;
    option.r = R;
    option.t = T;
    option.n = N;
    //	Print Option details
    printMultiOpt(&option);

    //	Cholevski factorization
    option.p = Chol(&p[0][0], option.n);

    // CPU Monte Carlo
    printf("\nMonte Carlo execution on CPU:\nN^ simulations: %d\n\n",SIMS);
    h_start = clock();
    CPU_sim=CPUBasketOptCall(&option, SIMS);
    h_stop = clock();
    CPU_timeSpent = ((float)(h_stop - h_start)) / CLOCKS_PER_SEC;
    
    price = CPU_sim.Expected;
    printf("Simulated price for the basket option: € %f with I.C [ %f;%f ]\n", price, price - CPU_sim.Confidence, price + CPU_sim.Confidence);
    printf("Total execution time: %f s\n\n", CPU_timeSpent);
    
    // GPU Monte Carlo
    printf("\nMonte Carlo execution on GPU:\nN^ simulations: %d\n",SIMS);
    CudaCheck( hipEventRecord( d_start, 0 ));
    GPUBasketOpt(&option, &GPU_sim);
    CudaCheck( hipEventRecord( d_stop, 0));
    CudaCheck( hipEventSynchronize( d_stop ));
    CudaCheck( hipEventElapsedTime( &GPU_timeSpent, d_start, d_stop ));
    GPU_timeSpent /= CLOCKS_PER_SEC;
    
    price = GPU_sim.Expected;
    printf("Simulated price for the basket option: € %f with I.C [ %f;%f ]\n", price, price-GPU_sim.Confidence, price + GPU_sim.Confidence);
    printf("Total execution time: %f s\n\n", GPU_timeSpent);
    
    // Comparing time spent with the two methods
    printf( "-\tComparing results:\t-\n");
    speedup = abs(CPU_timeSpent / GPU_timeSpent);
    printf( "The GPU's speedup: %.2f \n", speedup);
    return 0;
}


double* Chol( double *c, int n ){
    int i,j,k;
    double *a=(double*)malloc(n*n*sizeof(double));
    double v[n];
    for( i=0; i<n; i++){
        for( j=0; j<n; j++ ){
            if( j>=i ){
                //Triangolare inferiore
            	//v[j]=c[j][i]
                v[j] = c[i+j*n];
                for(k=0; k<i; k++)    //Scorre tutta
                    //v[j] = v[j] - a[i][k] * a[j][k]
                    v[j] = v[j]-(a[k+i*n] * a[k+j*n]);
                //a[j][i] = v[j] / sqrt( v[i] )
                if(v[i]>0)
                    a[i+j*n] = v[j]/sqrt( v[i] );
                else
                    a[i+j*n] = 0.0f;
            }
            else
                //Triangolare superiore a[j][i]
                a[i+j*n] = 0.0f;
        }
    }
    return a;
}
