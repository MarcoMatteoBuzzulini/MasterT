#include "hip/hip_runtime.h"
//
//  MonteCarlo.cu
//  tesi
//
//  Created by Marco Matteo Buzzulini on 27/11/17.
//  Copyright © 2017 Marco Matteo Buzzulini. All rights reserved.
//

#include "MonteCarlo.h"
#include <hip/hip_runtime.h>

// includes, project
#include <helper_functions.h> // Helper functions (utilities, parsing, timing)
#include <hip/hip_runtime_api.h>      // helper functions (cuda error checking and initialization)
#include <multithreading.h>

//	Host Black & Scholes
extern "C" double host_bsCall ( OptionData );

//	Host MonteCarlo
extern "C" OptionValue host_vanillaOpt(OptionData, int);

//	Device MonteCarlo
extern "C" OptionValue dev_vanillaOpt(OptionData *, int, int);

//	CVA: per ora è in test la simulazione delle Expected Exposures
extern "C" void dev_cvaEquityOption(OptionValue*, OptionData, CreditData, int, int, int);


///////////////////////////////////
//	PRINT FUNCTIONS
///////////////////////////////////

void printOption( OptionData o){
    printf("\n-\tOption data\t-\n\n");
    printf("Underlying asset price:\t € %.2f\n", o.s);
    printf("Strike price:\t\t € %.2f\n", o.k);
    printf("Risk free interest rate: %.2f\n", o.r);
    printf("Volatility:\t\t %.2f\n", o.v);
    printf("Time to maturity:\t %.2f %s\n", o.t, (o.t>1)?("years"):("year"));
}


///////////////////////////////////
//	ADJUST FUNCTIONS
///////////////////////////////////

void sizeAdjust(hipDeviceProp_t *deviceProp, int *numBlocks, int *numThreads){
	int maxGridSize = deviceProp->maxGridSize[0];
	int maxBlockSize = deviceProp->maxThreadsPerBlock;
	//	Replacing in case of wrong size
	if(*numBlocks > maxGridSize){
		*numBlocks = maxGridSize;
		printf("Warning: maximum size of Grid is %d",*numBlocks);
	}
	if(*numThreads > maxBlockSize){
		*numThreads = maxBlockSize;
		printf("Warning: maximum size of Blocks is %d",*numThreads);
	}
}

void memAdjust(hipDeviceProp_t *deviceProp, int *numThreads){
		size_t maxShared = deviceProp->sharedMemPerBlock;
		size_t maxConstant = deviceProp->totalConstMem;
		int sizeDouble = sizeof(double);
		int numShared = sizeDouble * *numThreads * 2;
		if(sizeof(MultiOptionData) > maxConstant){
			printf("\nWarning: Excess use of constant memory: %zu\n",maxConstant);
			printf("A double variable size is: %d\n",sizeDouble);
			printf("In a MultiOptionData struct there's a consumption of %zu constant memory\n",sizeof(MultiOptionData));
			printf("In this Basket Option there's %d stocks\n",N);
			int maxDim = (int)maxConstant/(sizeDouble*5);
			printf("The optimal number of dims should be: %d stocks\n",maxDim);
		}
		if(numShared > maxShared){
			printf("\nWarning: Excess use of shared memory: %zu\n",maxShared);
			printf("A double variable size is: %d\n",sizeDouble);
			int maxThreads = (int)maxShared / (2*sizeDouble);
			printf("The optimal number of thread should be: %d\n",maxThreads);
		}
		printf("\n");
}

void optimalAdjust(hipDeviceProp_t *deviceProp, int *numBlocks, int *numThreads){
	int multiProcessors = deviceProp->multiProcessorCount;
	int cudaCoresPM = _ConvertSMVer2Cores(deviceProp->major, deviceProp->minor);
	*numBlocks = multiProcessors * 40;
	*numThreads = pow(2,(int)(log(cudaCoresPM)/log(2)));
	sizeAdjust(deviceProp,numBlocks, numThreads);
}

void choseParameters(int *numBlocks, int *numThreads){
		hipDeviceProp_t deviceProp;
		CudaCheck(hipGetDeviceProperties(&deviceProp, 0));
		char risp;
		printf("\nParametri CUDA:\n");
		printf("Scegli il numero di Blocchi: ");
		scanf("%d",numBlocks);
		printf("Scegli il numero di Threads per blocco: ");
		scanf("%d",numThreads);
		printf("Vuoi ottimizzare i parametri inseriti? (Y/N) ");
		scanf("%s",&risp);
		if((risp=='Y')||(risp=='y'))
			optimalAdjust(&deviceProp,numBlocks, numThreads);
		else
			sizeAdjust(&deviceProp,numBlocks, numThreads);
		memAdjust(&deviceProp,numThreads);
}

////////////////////////////////////////////////////////////////////////////////////////
//                                      MAIN
////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, const char * argv[]) {
    /*--------------------------- DATA INSTRUCTION -----------------------------------*/
	OptionData option;
	option.v = 0.25;
	option.s = 100;
	option.k= 100.f;
	option.r= 0.05;
	option.t= 1.f;

	CreditData credit = {0,0,0};

	printf("Expected Exposures of an Equity Option\n");

	//	Definizione dei parametri CUDA per l'esecuzione in parallelo
	int numBlocks, numThreads;
	choseParameters(&numBlocks, &numThreads);

	printf("Simulazione di ( %d ; %d )\n",numBlocks, numThreads);
	int SIMS = numBlocks*PATH;

	//	Print Option details
	printOption(option);

	// PARAMETRI PER LA SIMULAZIONE EE
	// Scelta da tastiera del numero di simulazioni: di default 40
	int n = 40, i;
	double dt = option.t/(double)n;

    /*---------------- CORE COMPUTATIONS ----------------*/
	// Puntatore al vettore di prezzi simulati
    OptionValue *GPU_sim = (OptionValue *)malloc(sizeof(OptionValue)*n);
    
    float CPU_timeSpent=0, GPU_timeSpent=0, speedup;
    double *price = (double*)malloc(sizeof(double)*n);
    double *bs_price = (double*)malloc(sizeof(double)*n);
    double difference;

    clock_t h_start, h_stop;
    hipEvent_t d_start, d_stop;
    CudaCheck( hipEventCreate( &d_start ));
    CudaCheck( hipEventCreate( &d_stop ));

    //	Black & Scholes price
    for(i=0;i<n;i++){
    	bs_price[i] = host_bsCall(option);
    	option.t -= dt;
    }

    //	Ripristino valore originale del Time to mat
    option.t= 1.f;

   	printf("\nPrezzi Black & Scholes:\n");
   	printf("|\ti\t|\tPrezzo\t|\n");
   	for(i=0;i<n;i++)
   		printf("|\t%d\t|\t%f\t|\n",i,bs_price[i]);

    // CPU Monte Carlo
    /*
    printf("\nMonte Carlo execution on CPU:\nN^ simulations: %d\n\n",SIMS);
    h_start = clock();
    CPU_sim=host_vanillaOpt(option, SIMS);
    h_stop = clock();
    CPU_timeSpent = ((float)(h_stop - h_start)) / CLOCKS_PER_SEC;
    
    price = CPU_sim.Expected;
    printf("Simulated price for the basket option: € %f with I.C [ %f;%f ]\n", price, price - CPU_sim.Confidence, price + CPU_sim.Confidence);
    printf("Total execution time: %f s\n\n", CPU_timeSpent);
     */

    // GPU Monte Carlo
    printf("\nMonte Carlo execution on GPU:\nN^ simulations: %d\n",SIMS);
    CudaCheck( hipEventRecord( d_start, 0 ));
    dev_cvaEquityOption(GPU_sim, option, credit, n, numBlocks, numThreads);
    CudaCheck( hipEventRecord( d_stop, 0));
    CudaCheck( hipEventSynchronize( d_stop ));
    CudaCheck( hipEventElapsedTime( &GPU_timeSpent, d_start, d_stop ));
    GPU_timeSpent /= 1000;
    
    printf("\nPrezzi Simulati:\n");
   	printf("|\ti\t|\Differenza di prezzo\t|\n");
   	for(i=0;i<n;i++)
   		printf("|\t%d\t|\t%f\t|\n",i,GPU_sim[i].Expected);

    printf("Total execution time: %f s\n\n", GPU_timeSpent);
    
    // Comparing time spent with the two methods
    printf( "-\tComparing results:\t-\n");
    printf("\nDifferenza Prezzi:\n");
  	printf("|\ti\t|\tPrezzo\t|");
  	for(i=0;i<n;i++){
  		difference = abs(GPU_sim[i].Expected - bs_price[i]);
   		printf("|\t%d\t|\t%f\t|\n",i,difference);
  	}

    return 0;
}
