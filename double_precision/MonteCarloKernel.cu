#include "hip/hip_runtime.h"
/*
 *  MonteCarloKernel.cu
 *  Monte Carlo methods in CUDA
 *  Dissertation project
 *  Created on: 06/feb/2018
 *  Author: Marco Matteo Buzzulini
 */

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "MonteCarlo.h"
#define max(a,b) \
({ __typeof__ (a) _a = (a); \
__typeof__ (b) _b = (b); \
_a > _b ? _a : _b; })

// Struct for Monte Carlo methods
typedef struct{
    OptionValue *h_CallValue, *d_CallValue;
    OptionValue callValue;
    OptionData sopt;
    MultiOptionData mopt;
    hiprandState *RNG;
    int numBlocks, numThreads, numOpt, path;
} dev_MonteCarloData;

// Memory initialization for MC
void MonteCarlo_init(dev_MonteCarloData *data);
// Freeing memory after MC
void MonteCarlo_closing(dev_MonteCarloData *data);
// Monte Carlo method for Option Pricing
void MonteCarlo(dev_MonteCarloData *data);
// Monte Carlo method for CVA - 1 black-scholes option
void cvaMonteCarlo(dev_MonteCarloData *data, double intdef, double lgd, int n_grid);

////////////////////////////////////////////////////////////////
////////////////    CONSTANT MEMORY     ////////////////////////
////////////////////////////////////////////////////////////////

// Basket Option
__device__ __constant__ MultiOptionData MOPTION;
// Vanilla Call Option
__device__ __constant__ OptionData OPTION;
// Number of underlyings, num simulations per block and the sims for CVA
__device__ __constant__ int N_OPTION, N_PATH, N_GRID;
// Financial parameters for CVA: Default intensity and Loss given default
__device__ __constant__ double INTDEF, LGD;

////////////////////////////////////////////////////////////////
////////////////    KERNEL FUNCTIONS    ////////////////////////
////////////////////////////////////////////////////////////////

/*  *   *   *   *   ONLY DEVICE   *   *   *   *   */
// Call Option payoff
__device__ double callPayoff(hiprandState *threadState){
    double z = hiprand_normal(threadState);
    double s = OPTION.s * exp((OPTION.r - 0.5 * OPTION.v * OPTION.v) * OPTION.t + OPTION.v * sqrt(OPTION.t) * z);
    return max(s - OPTION.k,0);
}

// Basket option random number
__device__ void brownianVect(double *bt, hiprandState *threadState){
    int i,j;
    double g[N];
    for(i=0;i<N_OPTION;i++)
        g[i]=hiprand_normal(threadState);
    for(i=0;i<N_OPTION;i++){
        double somma = 0;
        for(j=0;j<N_OPTION;j++)
            somma += MOPTION.p[i][j] * g[j];
        bt[i] = somma;
    }
    for(i=0;i<N_OPTION;i++)
        bt[i] += MOPTION.d[i];
}
// Basket option payoff
__device__ double basketPayoff(double *bt){
    int j;
    double s[N], st_sum=0, price;
    for(j=0;j<N_OPTION;j++)
        s[j] = MOPTION.s[j] * exp((MOPTION.r - 0.5 * MOPTION.v[j] * MOPTION.v[j])*MOPTION.t+MOPTION.v[j] * bt[j] * sqrt(MOPTION.t));
    // Third step: Mean price
    for(j=0;j<N_OPTION;j++)
        st_sum += s[j] * MOPTION.w[j];
    // Fourth step: Option payoff
    price = st_sum - MOPTION.k;
    
    return max(price,0);
}

// Simulating Geometric Brownian path
__device__ double geomBrownian( double s, double t, double z ){
    double x = (OPTION.r - 0.5 * OPTION.v * OPTION.v) * t + OPTION.v * sqrt(t) * z;
    return s * exp(x);
}

// Hastings approximation of cumulative normal distribution
__device__ double cnd(double d){
    const double       A1 = 0.31938153;
    const double       A2 = -0.356563782;
    const double       A3 = 1.781477937;
    const double       A4 = -1.821255978;
    const double       A5 = 1.330274429;
    const double ONEOVER2PI = 0.39894228040143267793994605993438;
    double K = 1.0 / (1.0 + 0.2316419 * fabs(d));
    double cnd = ONEOVER2PI * exp(- 0.5 * d * d) * (K * (A1 + K * (A2 + K * (A3 + K * (A4 + K * A5)))));
    if (d > 0)
        return 1.0 - cnd;
    else
        return cnd;
}
// Black & Scholes price formula for vanilla options
__device__ double device_bsCall ( double s, double t){
    double d1 = ( log(s / OPTION.k) + (OPTION.r + 0.5 * OPTION.v * OPTION.v) * t) / (OPTION.v * sqrt(t));
    double d2 = d1 - OPTION.v * sqrt(t);
    return s * cnd(d1) - OPTION.k * exp(- OPTION.r * t) * cnd(d2);
}

/*  *   *   *   *   GLOBAL  *   *   *   *   */
// Basket Option Kernel
__global__ void basketOptMonteCarlo(hiprandState * randseed, OptionValue *d_CallValue){
    // Parameters for shared memory
    int sumIndex = threadIdx.x;
    int sum2Index = sumIndex + blockDim.x;
    /*  - SHARED MEMORY -  */
    extern __shared__ double s_Sum[];
    
    // Global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Copy random number state to local memory
    hiprandState threadState = randseed[tid];
    int i;
    OptionValue sum = {0, 0};
    
    for( i=sumIndex; i<N_PATH; i+=blockDim.x){
        double price=0.0f, bt[N];
        // Random Number Generation
        brownianVect(bt,&threadState);
        // Price simulation with the basket call option payoff function
        price=basketPayoff(bt);
        // Mean sum
        sum.Expected += price;
        sum.Confidence += price*price;
    }
    // Copy to the shared memory
    s_Sum[sumIndex] = sum.Expected;
    s_Sum[sum2Index] = sum.Confidence;
    __syncthreads();
    // Reduce shared memory accumulators and write final result to global memory
    int halfblock = blockDim.x/2;
    // Reduction in log2(threadBlocks) steps, so threadBlock must be power of 2
    do{
        if ( sumIndex < halfblock ){
            s_Sum[sumIndex] += s_Sum[sumIndex+halfblock];
            s_Sum[sum2Index] += s_Sum[sum2Index+halfblock];
        }
        __syncthreads();
        halfblock /= 2;
    }while ( halfblock != 0 );
    // Copy to the global memory
    if (sumIndex == 0){
        d_CallValue[blockIdx.x].Expected = s_Sum[sumIndex];
        d_CallValue[blockIdx.x].Confidence = s_Sum[sum2Index];
    }
}
// Vanilla Option call Kernel
__global__ void vanillaOptMonteCarlo(hiprandState * randseed, OptionValue *d_CallValue){
    // Parameters for shared memory
    int sumIndex = threadIdx.x;
    int sum2Index = sumIndex + blockDim.x;
    /*  - SHARED MEMORY -  */
    extern __shared__ double s_Sum[];
    
    // Global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Copy random number state to local memory
    hiprandState threadState = randseed[tid];
    
    OptionValue sum = {0, 0};
    int i;
    for( i=sumIndex; i<N_PATH; i+=blockDim.x){
        double price=0.0f;
        // Price simulation with the vanilla call option payoff function
        price = callPayoff(&threadState);
        sum.Expected += price;
        sum.Confidence += price*price;
    }
    // Copy to the shared memory
    s_Sum[sumIndex] = sum.Expected;
    s_Sum[sum2Index] = sum.Confidence;
    __syncthreads();
    // Reduce shared memory accumulators and write final result to global memory
    int halfblock = blockDim.x/2;
    // Reduction in log2(threadBlocks) steps, so threadBlock must be power of 2
    do{
        if ( sumIndex < halfblock ){
            s_Sum[sumIndex] += s_Sum[sumIndex+halfblock];
            s_Sum[sum2Index] += s_Sum[sum2Index+halfblock];
        }
        __syncthreads();
        halfblock /= 2;
    }while ( halfblock != 0 );
    // Copy to the global memory
    if (sumIndex == 0){
        d_CallValue[blockIdx.x].Expected = s_Sum[sumIndex];
        d_CallValue[blockIdx.x].Confidence = s_Sum[sum2Index];
    }
}

__global__ void cvaCallOptMC(hiprandState * randseed, OptionValue *d_CallValue){
    // Parameters for shared memory
    int sumIndex = threadIdx.x;
    int sum2Index = sumIndex + blockDim.x;
    /*  - SHARED MEMORY -  */
    extern __shared__ double s_Sum[];
    
    // Global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Copy random number state to local memory
    hiprandState threadState = randseed[tid];
    
    double dt = OPTION.t / N_GRID;
    // Calcolo di un CVA
    // Step 1: simulare traiettoria sottostante, ad ogni istante dt calcolare prezzo opzione attualizzato con B&S
    // Step 2: calcolo CVA per ogni traiettoria e sommarlo alla variabile mean_price
    // Step 3: salvare nella memoria condivisa i CVA calcolati
    OptionValue sum = {0, 0};
    int i,j;
    for( i=sumIndex; i<N_PATH; i+=blockDim.x){
        double s, ee, t;
        double mean_price = 0;
        s = OPTION.s;
        t = OPTION.t;
        ee = device_bsCall(s,t);
        for(j=1; j <= N_GRID; j++){
            double dp = exp(-(dt*(j-1)) * INTDEF) - exp(-(dt*j) * INTDEF);
            if( (t -= dt)>=0 ){
                double z = hiprand_normal(&threadState);
                s = geomBrownian(s, dt, z);
                ee = device_bsCall(s,t);
            }
            else{
                ee = 0;
            }
            mean_price += dp * ee;
        }
        mean_price *= LGD;
        sum.Expected += mean_price;
        sum.Confidence += mean_price * mean_price;
    }
    // Copy to the shared memory
    s_Sum[sumIndex] = sum.Expected;
    s_Sum[sum2Index] = sum.Confidence;
    __syncthreads();
    // Reduce shared memory accumulators and write final result to global memory
    int halfblock = blockDim.x/2;
    // Reduction in log2(threadBlocks) steps, so threadBlock must be power of 2
    do{
        if ( sumIndex < halfblock ){
            s_Sum[sumIndex] += s_Sum[sumIndex+halfblock];
            s_Sum[sum2Index] += s_Sum[sum2Index+halfblock];
        }
        __syncthreads();
        halfblock /= 2;
    }while ( halfblock != 0 );
    // Copy to the global memory
    if (sumIndex == 0){
        d_CallValue[blockIdx.x].Expected = s_Sum[sumIndex];
        d_CallValue[blockIdx.x].Confidence = s_Sum[sum2Index];
    }
}

__global__ void randomSetup( hiprandState *randSeed ){
    // Global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Each thread block gets different seed, threads within a thread block get different sequence numbers
    hiprand_init(blockIdx.x + gridDim.x, threadIdx.x, 0, &randSeed[tid]);
}

////////////////////////////////////////////////////////////////
////////////////    HOST FUNCTIONS  ////////////////////////////
////////////////////////////////////////////////////////////////

void MonteCarlo_init(dev_MonteCarloData *data){
    hipEvent_t start, stop;
    CudaCheck( hipEventCreate( &start ));
    CudaCheck( hipEventCreate( &stop ));
    float time;
    
    /*--------------- CONSTANT MEMORY ----------------*/
    if( data->numOpt > 1){
        int n_option = data->numOpt;
        CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(N_OPTION),&n_option,sizeof(int)));
    }
    
    int n_path = data->path;
    printf("Numero di simulazioni per blocco: \t %d\n",n_path);
    printf("Numero di simulazioni per processo: \t %d\n",n_path/data->numThreads);
    CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(N_PATH),&n_path,sizeof(int)));
    
    // RANDOM NUMBER GENERATION KERNEL
    //Allocate states for pseudo random number generators
    CudaCheck(hipMalloc((void **) &data->RNG, data->numBlocks * data->numThreads * sizeof(hiprandState)));
    //Setup for the random number sequence
    CudaCheck( hipEventRecord( start, 0 ));
    randomSetup<<<data->numBlocks, data->numThreads>>>(data->RNG);
    cuda_error_check("\Errore nel lancio randomSetup: ","\n");
    CudaCheck( hipEventRecord( stop, 0));
    CudaCheck( hipEventSynchronize( stop ));
    CudaCheck( hipEventElapsedTime( &time, start, stop ));
    printf( "RNG done in ms \t %f\n", time);
    
    //    Host Memory Allocation
    CudaCheck( hipEventRecord( start, 0 ));
    CudaCheck(hipHostMalloc(&data->h_CallValue, sizeof(OptionValue)*data->numBlocks));
    CudaCheck( hipEventRecord( stop, 0));
    CudaCheck( hipEventSynchronize( stop ));
    CudaCheck( hipEventElapsedTime( &time, start, stop ));
    printf( "Host memory allocation done in ms \t %f\n", time);
    //    Device Memory Allocation
    CudaCheck( hipEventRecord( start, 0 ));
    CudaCheck(hipMalloc(&data->d_CallValue, sizeof(OptionValue)*data->numBlocks));
    CudaCheck( hipEventRecord( stop, 0));
    CudaCheck( hipEventSynchronize( stop ));
    CudaCheck( hipEventElapsedTime( &time, start, stop ));
    printf( "Device memory allocation done in ms \t %f\n", time);
    
    CudaCheck( hipEventDestroy( start ));
    CudaCheck( hipEventDestroy( stop ));
}

void MonteCarlo_closing(dev_MonteCarloData *data){
    hipEvent_t start, stop;
    CudaCheck( hipEventCreate( &start ));
    CudaCheck( hipEventCreate( &stop ));
    float time;
    
    CudaCheck( hipEventRecord( start, 0 ));
    //Free memory space
    CudaCheck(hipFree(data->RNG));
    CudaCheck(hipHostFree(data->h_CallValue));
    CudaCheck(hipFree(data->d_CallValue));
    
    CudaCheck( hipEventRecord( stop, 0));
    CudaCheck( hipEventSynchronize( stop ));
    CudaCheck( hipEventElapsedTime( &time, start, stop ));
    printf( "Free memory done in ms \t %f\n", time);
    
    CudaCheck( hipEventDestroy( start ));
    CudaCheck( hipEventDestroy( stop ));
}

void MonteCarlo(dev_MonteCarloData *data){
    hipEvent_t start, stop;
    CudaCheck( hipEventCreate( &start ));
    CudaCheck( hipEventCreate( &stop ));
    float time, r,t;
    
    /*----------------- SHARED MEMORY -------------------*/
    int i, numShared = sizeof(double) * data->numThreads * 2;
    
    /*--------------- CONSTANT MEMORY ----------------*/
    if( data->numOpt == 1){
        r = data->sopt.r;
        t = data->sopt.t;
        CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(OPTION),&data->sopt,sizeof(OptionData)));
        // Time
        CudaCheck( hipEventRecord( start, 0 ));
        vanillaOptMonteCarlo<<<data->numBlocks, data->numThreads, numShared>>>(data->RNG,(OptionValue *)(data->d_CallValue));
        cuda_error_check("\Errore nel lancio vanillaOptMonteCarlo: ","\n");
        CudaCheck( hipEventRecord( stop, 0));
        CudaCheck( hipEventSynchronize( stop ));
        CudaCheck( hipEventElapsedTime( &time, start, stop ));
        printf( "Kernel done in ms \t %f\n", time);
    }
    else{
        r = data->mopt.r;
        t = data->mopt.t;
        CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(MOPTION),&data->mopt,sizeof(MultiOptionData)));
        // Time
        CudaCheck( hipEventRecord( start, 0 ));
        basketOptMonteCarlo<<<data->numBlocks, data->numThreads, numShared>>>(data->RNG,(OptionValue *)(data->d_CallValue));
        cuda_error_check("\Errore nel lancio basketOptMonteCarlo: ","\n");
        CudaCheck( hipEventRecord( stop, 0));
        CudaCheck( hipEventSynchronize( stop ));
        CudaCheck( hipEventElapsedTime( &time, start, stop ));
        printf( "Kernel done in ms \t %f\n", time);
    }
    
    //MEMORY CPY: prices per block
    // Time
    CudaCheck( hipEventRecord( start, 0 ));
    CudaCheck(hipMemcpy(data->h_CallValue, data->d_CallValue, data->numBlocks * sizeof(OptionValue), hipMemcpyDeviceToHost));
    CudaCheck( hipEventRecord( stop, 0));
    CudaCheck( hipEventSynchronize( stop ));
    CudaCheck( hipEventElapsedTime( &time, start, stop ));
    printf( "Copy from device-to-host done in ms \t %f\n", time);
    
    // Closing Monte Carlo
    double sum=0, sum2=0, price, empstd;
    long int nSim = data->numBlocks * data->path;
    // Time
    CudaCheck( hipEventRecord( start, 0 ));
    for ( i = 0; i < data->numBlocks; i++ ){
        sum += data->h_CallValue[i].Expected;
        sum2 += data->h_CallValue[i].Confidence;
    }
    price = exp(-r*t) * (sum/(double)nSim);
    empstd = sqrt((double)((double)nSim * sum2 - sum * sum)/((double)nSim * (double)(nSim - 1)));
    data->callValue.Confidence = 1.96 * empstd / (double)sqrt((double)nSim);
    data->callValue.Expected = price;
    CudaCheck( hipEventRecord( stop, 0));
    CudaCheck( hipEventSynchronize( stop ));
    CudaCheck( hipEventElapsedTime( &time, start, stop ));
    printf( "Call price done in ms \t %f\n", time);
    
    CudaCheck( hipEventDestroy( start ));
    CudaCheck( hipEventDestroy( stop ));
}

void cvaMonteCarlo(dev_MonteCarloData *data, double intdef, double lgd, int n_grid){
    hipEvent_t start, stop;
    CudaCheck( hipEventCreate( &start ));
    CudaCheck( hipEventCreate( &stop ));
    float time;
    
    /*----------------- SHARED MEMORY -------------------*/
    int i, numShared = sizeof(double) * data->numThreads * 2;
    /*--------------- CONSTANT MEMORY ----------------*/
    CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(INTDEF), &intdef, sizeof(double)));
    CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(LGD), &lgd, sizeof(double)));
    CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(N_GRID), &n_grid, sizeof(int)));
    CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(OPTION), &data->sopt, sizeof(OptionData)));
    //Time
    CudaCheck( hipEventRecord( start, 0 ));
    cvaCallOptMC<<<data->numBlocks, data->numThreads, numShared>>>(data->RNG,(OptionValue *)(data->d_CallValue));
    cuda_error_check("\Errore nel lancio cvaCallOptMC: ","\n");
    CudaCheck( hipEventRecord( stop, 0));
    CudaCheck( hipEventSynchronize( stop ));
    CudaCheck( hipEventElapsedTime( &time, start, stop ));
    printf( "Kernel done in ms \t %f\n", time);
    
    //MEMORY CPY: prices per block
    CudaCheck(hipMemcpy(data->h_CallValue, data->d_CallValue, data->numBlocks * sizeof(OptionValue), hipMemcpyDeviceToHost));
    
    // Closing Monte Carlo
    double sum=0, sum2=0, price, empstd;
    long int nSim = data->numBlocks * data->path;
    CudaCheck( hipEventRecord( start, 0 ));
    for ( i = 0; i < data->numBlocks; i++ ){
        sum += data->h_CallValue[i].Expected;
        sum2 += data->h_CallValue[i].Confidence;
    }
    price = sum/(double)nSim;
    empstd = sqrt((double)((double)nSim * sum2 - sum * sum)/((double)nSim * (double)(nSim - 1)));
    data->callValue.Confidence = 1.96 * empstd / (double)sqrt((double)nSim);
    data->callValue.Expected = price;
    CudaCheck( hipEventRecord( stop, 0));
    CudaCheck( hipEventSynchronize( stop ));
    CudaCheck( hipEventElapsedTime( &time, start, stop ));
    printf( "CVA price done in ms \t %f\n", time);
    
    CudaCheck( hipEventDestroy( start ));
    CudaCheck( hipEventDestroy( stop ));
}

////////////////////////////////////////////////
////////////////    WRAPPERS    ////////////////
////////////////////////////////////////////////

extern "C" OptionValue dev_basketOpt(MultiOptionData *option, int numBlocks, int numThreads, int sims){
    dev_MonteCarloData data;
    // Option
    data.mopt = *option;
    // Kernel parameters
    data.numBlocks = numBlocks;
    data.numThreads = numThreads;
    data.numOpt = N;
    data.path = sims / numBlocks;
    // Core
    MonteCarlo_init(&data);
    MonteCarlo(&data);
    MonteCarlo_closing(&data);
    
    return data.callValue;
}

extern "C" OptionValue dev_vanillaOpt(OptionData *opt, int numBlocks, int numThreads, int sims){
    dev_MonteCarloData data;
    // Option
    data.sopt = *opt;
    // Kernel parameters
    data.numBlocks = numBlocks;
    data.numThreads = numThreads;
    data.numOpt = 1;
    data.path = sims / numBlocks;
    // Core
    MonteCarlo_init(&data);
    MonteCarlo(&data);
    MonteCarlo_closing(&data);
    
    return data.callValue;
}

extern "C" OptionValue dev_cvaEquityOption(CVA *cva, int numBlocks, int numThreads, int sims){
    dev_MonteCarloData data;
    data.sopt = cva->option;
    // Kernel parameters
    data.numBlocks = numBlocks;
    data.numThreads = numThreads;
    data.numOpt = 1;
    data.path = sims / numBlocks;
    // Core
    MonteCarlo_init(&data);
    cvaMonteCarlo(&data, (double)cva->defInt, (double)cva->lgd, cva->n);
    // Closing
    MonteCarlo_closing(&data);
    
    return data.callValue;
}


