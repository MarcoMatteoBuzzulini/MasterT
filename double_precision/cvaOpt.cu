#include "hip/hip_runtime.h"
//
//  MonteCarlo.cu
//  tesi
//
//  Created by Marco Matteo Buzzulini on 27/11/17.
//  Copyright © 2017 Marco Matteo Buzzulini. All rights reserved.
//

#include "MonteCarlo.h"

#define THREADS 256
#define BLOCKS 512
#define PATH 40
#define SIMPB 131072

extern "C" double host_bsCall ( OptionData );
extern "C" void host_cvaEquityOption(CVA *, int);
extern "C" OptionValue dev_cvaEquityOption(CVA *, int, int, int);
extern "C" void printOption( OptionData o);
extern "C" void Chol( double c[N][N], double a[N][N] );
extern "C" void printMultiOpt( MultiOptionData *o);
extern "C" double randMinMax(double min, double max);

void getRandomSigma( double* std );
void getRandomRho( double* rho );
void pushVett( double* vet, double x );

const double defInt = 0.03;
const double recoveryRate = 0.4;
const double S = 100;
const double K = 100;
const double R = 0.05;
const double V = 0.2;
const double T = 1.f;

int main(int argc, const char * argv[]) {
    /*--------------------------- DATA INSTRUCTION -----------------------------------*/
    CVA cva;
    cva.defInt = defInt;
    cva.lgd = (1 - recoveryRate);
    cva.n = PATH;
    
    // n+1 because it starts from 1
    double *bs_price = (double*)malloc(sizeof(double)*(cva.n+1));

    OptionValue result;
    printf("\nCVA of an European call Option\nIntensita di default %.2f, LGD %.2f\n",cva.defInt,cva.lgd);
    cva.opt.v = V;
    cva.opt.s = S;
    cva.opt.t = T;
    cva.opt.r = R;
    cva.opt.k = K;
    cva.ns = 1;
    
    hipEvent_t d_start, d_stop;
    int i, j, SIMS;
    double dt, cholRho[N][N];
    float GPU_timeSpent=0;
    
	//	CUDA Parameters optimized
    printf("Inserisci il numero di simulazioni Monte Carlo(x131.072): ");
    scanf("%d",&SIMS);
    SIMS *= SIMPB;
    printf("\nScenari di Monte Carlo: %d\n",SIMS);
    
    printOption(cva.option);
    bs_price[0] = host_bsCall(cva.option);
    int n = cva.option.t;
    dt = cva.option.t/(double)cva.n;
    for(i=1;i<cva.n+1;i++){
        if((cva.option.t -= dt)<0)
            bs_price[i] = 0;
        else
            bs_price[i] = host_bsCall(cva.option);
    }
    cva.option.t = n;
    

	// Timer init
    CudaCheck( hipEventCreate( &d_start ));
    CudaCheck( hipEventCreate( &d_stop ));
    
    // CPU Monte Carlo
    /*
    printf("\nCVA execution on CPU:\n");
    CudaCheck( hipEventRecord( d_start, 0 ));
    host_cvaEquityOption(&cva, SIMS);
    CudaCheck( hipEventRecord( d_stop, 0));
    CudaCheck( hipEventSynchronize( d_stop ));
    CudaCheck( hipEventElapsedTime( &CPU_timeSpent, d_start, d_stop ));
    CPU_timeSpent /= 1000;
    printf("\nPrezzi Simulati:\n");
    
        printf("|\ti\t\t|\tPrezzi BS\t| Differenza Prezzi\t|\tPrezzi\t\t|\tDefault Prob\t|\n");
        for(i=0;i<cva.n+1;i++){
            difference = abs(cva.ee[i].Expected - bs_price[i]);
            printf("|\t%f\t|\t%f\t|\t%f\t|\t%f\t|\t%f\t|\n",dt*i,bs_price[i],difference,cva.ee[i].Expected,cva.dp[i]);
        }
    
    printf("\nCVA: %f\n\n",cva.cva);
    printf("\nTotal execution time: %f s\n\n", CPU_timeSpent);
    printf("--------------------------------------------------\n");
     */
    // GPU Monte Carlo
    printf("\nCVA execution on GPU:\n");
    CudaCheck( hipEventRecord( d_start, 0 ));
    result = dev_cvaEquityOption(&cva, BLOCKS, THREADS, SIMS);
    CudaCheck( hipEventRecord( d_stop, 0));
    CudaCheck( hipEventSynchronize( d_stop ));
    CudaCheck( hipEventElapsedTime( &GPU_timeSpent, d_start, d_stop ));
    //GPU_timeSpent /= 1000;

    printf("\nCVA: %f\n\n",cva.cva);
    //printf("Speed up: %f\n\n",CPU_timeSpent/GPU_timeSpent);
   	free(bs_price);
    return 0;
}

