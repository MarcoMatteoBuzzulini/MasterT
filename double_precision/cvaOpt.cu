//
//  MonteCarlo.cu
//  tesi
//
//  Created by Marco Matteo Buzzulini on 27/11/17.
//  Copyright © 2017 Marco Matteo Buzzulini. All rights reserved.
//

#include "MonteCarlo.h"

#define THREADS 128
#define BLOCKS 512
#define PATH 100
#define SIMPB 131072

extern "C" OptionValue host_cvaEquityOption(CVA *, int);
extern "C" OptionValue dev_cvaEquityOption(CVA *, int, int, int);

extern "C" void printOption(OptionData o);

const double defInt = 0.03;
const double recoveryRate = 0.4;
const double S = 100;
const double K = 100;
const double R = 0.05;
const double V = 0.2;
const double T = 1.f;

int main(int argc, const char * argv[]) {
    /*--------------------------- DATA INSTRUCTION -----------------------------------*/
    CVA cva;
    cva.defInt = defInt;
    cva.lgd = (1 - recoveryRate);
    cva.n = PATH;
    
    printf("\nCVA of an European call Option\nDefault intensity %.2f, LGD %.2f\n",cva.defInt,cva.lgd);
    cva.option.v = V;
    cva.option.s = S;
    cva.option.t = T;
    cva.option.r = R;
    cva.option.k = K;
    cva.ns = 1;
    
    hipEvent_t d_start, d_stop;
    int SIMS;
    float GPU_timeSpent=0, CPU_timeSpent=0;
    OptionValue dev_result = {0,0}, host_result = {0,0};
    
	//	CUDA Parameters optimized
    printf("Monte Carlo simulations(x131.072): ");
    scanf("%d",&SIMS);
    SIMS *= SIMPB;
    printf("\nTotal Monte Carlo paths: %d\n",SIMS);
    
    printOption(cva.option);

	// Timer init
    CudaCheck( hipEventCreate( &d_start ));
    CudaCheck( hipEventCreate( &d_stop ));
    
    // CPU Monte Carlo
    
    printf("\nCVA execution on CPU...\n");
    CudaCheck( hipEventRecord( d_start, 0 ));
    host_result = host_cvaEquityOption(&cva, SIMS);
    CudaCheck( hipEventRecord( d_stop, 0));
    CudaCheck( hipEventSynchronize( d_stop ));
    CudaCheck( hipEventElapsedTime( &CPU_timeSpent, d_start, d_stop ));
    
    printf("\nCVA: %f\nConfidence Interval: %f\n\n",host_result.Expected, host_result.Confidence);
    printf("\nTotal execution time: %f s\n\n", CPU_timeSpent);
    printf("--------------------------------------------------\n");
    
    // GPU Monte Carlo
    printf("\nCVA execution on GPU...\n");
    CudaCheck( hipEventRecord( d_start, 0 ));
    dev_result = dev_cvaEquityOption(&cva, BLOCKS, THREADS, SIMS);
    CudaCheck( hipEventRecord( d_stop, 0));
    CudaCheck( hipEventSynchronize( d_stop ));
    CudaCheck( hipEventElapsedTime( &GPU_timeSpent, d_start, d_stop ));

    printf("\nTotal execution time: %f ms\n\n", GPU_timeSpent);
    printf("\nCVA: %f\nConfidence Interval: %f\n\n",dev_result.Expected, dev_result.Confidence);
    printf("Speed up: %f\n\n",CPU_timeSpent/GPU_timeSpent);
    return 0;
}

