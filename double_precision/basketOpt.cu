#include "hip/hip_runtime.h"
/*
 *  basketOpt.cu
 *  Monte Carlo methods in CUDA
 *  Dissertation project
 *  Created on: 06/feb/2018
 *  Author: Marco Matteo Buzzulini
 *  Copyright © 2018 Marco Matteo Buzzulini. All rights reserved.
 */

#include "MonteCarlo.h"

#define NTHREADS 1
#define THREADS 128
#define BLOCKS 512
#define SIMPB 131072

extern "C" OptionValue host_basketOpt(MultiOptionData*, int);
extern "C" OptionValue dev_basketOpt(MultiOptionData *, int, int,int);
extern "C" void Chol( double c[N][N], double a[N][N] );
extern "C" void printMultiOpt( MultiOptionData *o);
extern "C" double randMinMax(double min, double max);

void getRandomSigma( double* std );
void getRandomRho( double* rho );
void pushVett( double* vet, double x );

int main(int argc, const char * argv[]) {
    /*--------------------------- VARIABLES -----------------------------------*/
	double dw = (double)1 / N;

	// Option Data
	MultiOptionData option;
	//	Volatility
	option.v[0] = 0.2;
	option.v[1] = 0.3;
	option.v[2] = 0.2;
	//	Spot prices
	option.s[0] = 100;
	option.s[1] = 100;
	option.s[2] = 100;
	//	Weights
	option.w[0] = dw;
	option.w[1] = dw;
	option.w[2] = dw;
	//	Correlations
	option.p[0][0] = 1;
			option.p[0][1] = -0.5;
					option.p[0][2] = -0.5;
	option.p[1][0] = -0.5;
			option.p[1][1] = 1;
					option.p[1][2] = -0.5;
	option.p[2][0] = -0.5;
			option.p[2][1] = -0.5;
					option.p[2][2] = 1;
	//	Drift vectors for the brownians
	option.d[0] = 0;
	option.d[1] = 0;
	option.d[2] = 0;
	option.k= 100.f;
	option.r= 0.048790164;
	option.t= 1.f;
	if(N!=3){
		getRandomSigma(option.v);
		getRandomRho(&option.p[0][0]);
		pushVett(option.s,100);
		pushVett(option.w,dw);
		pushVett(option.d,0);
	}

	// Simulation variables
	int numBlocks, numThreads[NTHREADS], SIMS, i, j;
	OptionValue CPU_sim, GPU_sim[NTHREADS];
	float CPU_timeSpent=0, GPU_timeSpent[NTHREADS], speedup[NTHREADS];
	double cholRho[N][N], difference[NTHREADS];
	// Timer
	hipEvent_t d_start, d_stop;

	/*--------------------- START PROGRAM ------------------------------*/
	printf("Basket Option Pricing\n");
	//	CUDA parameters for parallel execution
    numBlocks = BLOCKS;
    numThreads[0] = THREADS;
    //numThreads[1] = 128;
    //numThreads[2] = 1024;
    //numThreads[3] = 512;
    printf("Inserisci il numero simulazioni (x131.072): ");
    scanf("%d",&SIMS);
    SIMS *= SIMPB;
	printf("\nScenari di Monte Carlo: %d\n",SIMS);
	//	Print Option details
	if(N<7)
		printMultiOpt(&option);
	else
		printf("\nBasket Option con %d sottostanti\n",N);
    //	Cholevski factorization
    Chol(option.p, cholRho);
    for(i=0;i<N;i++)
    	for(j=0;j<N;j++)
           	option.p[i][j]=cholRho[i][j];
    // Timer init
    CudaCheck( hipEventCreate( &d_start ));
    CudaCheck( hipEventCreate( &d_stop ));
    /* CPU Monte Carlo */
    printf("\nMonte Carlo execution on CPU...\n");
    CudaCheck( hipEventRecord( d_start, 0 ));
    CPU_sim=host_basketOpt(&option, SIMS);
    CudaCheck( hipEventRecord( d_stop, 0));
    CudaCheck( hipEventSynchronize( d_stop ));
    CudaCheck( hipEventElapsedTime( &CPU_timeSpent, d_start, d_stop ));
    //CPU_timeSpent /= 1000;

    // GPU Monte Carlo
    printf("\nMonte Carlo execution on GPU...\n");
    for(i=0; i<NTHREADS; i++){
        printf("Monte Carlo for (%d,%d) x %d simulations per thread\n", BLOCKS, numThreads[i], SIMS/BLOCKS/numThreads[i]);
    	CudaCheck( hipEventRecord( d_start, 0 ));
       	GPU_sim[i] = dev_basketOpt(&option, numBlocks, numThreads[i], SIMS);
        CudaCheck( hipEventRecord( d_stop, 0));
        CudaCheck( hipEventSynchronize( d_stop ));
        CudaCheck( hipEventElapsedTime( &GPU_timeSpent[i], d_start, d_stop ));
        //GPU_timeSpent[i] /= 1000;
        difference[i] = abs(GPU_sim[i].Expected - CPU_sim.Expected);
        speedup[i] = abs(CPU_timeSpent / GPU_timeSpent[i]);
        printf("\n");
    }
    // Comparing time spent with the two methods
    printf( "\n-\tResults:\t-\n");
    printf("Simulated price for the option with CPU: Expected price, I.C., time\n%f \n%f \n%.1f \n", CPU_sim.Expected, CPU_sim.Confidence, CPU_timeSpent);
    printf("Simulated price for the option with GPU:\n");
    printf("  : NumThreads : Price : Confidence Interval : Difference from BS price :  Time : Speedup :");
    printf("\n");
    for(i=0; i<NTHREADS; i++){
        printf("%d \n",numThreads[i]);
        printf("%f \n",GPU_sim[i].Expected);
        printf("%f \n",GPU_sim[i].Confidence);
        printf("%f \n",difference[i]);
        printf("%.1f \n",GPU_timeSpent[i]);
        printf("%.2f \n",speedup[i]);
        printf("---\n");
    }
    CudaCheck( hipEventDestroy( d_start ));
    CudaCheck( hipEventDestroy( d_stop ));
    return 0;
}

//Simulation std, rho and covariance matrix
void getRandomSigma( double* std ){
    int i,j=0;
    for(i=0;i<N;i++){
        if(j==0){
            std[i]=0.3;
            j=1;
        }
        else{
            std[i]=0.2;
            j=0;
        }
    }
}
void getRandomRho( double* rho ){
    int i,j;
    //creating the vectors of rhos
    for(i=0;i<N;i++){
        for(j=i;j<N;j++){
            double r;
            if(i==j)
                r=1;
            else
                if(j%2==0)
                    r = 0.5;
                else
                    r= -0.5;
            rho[j+i*N] = r;
            rho[i+j*N] = r;
        }
    }
}
void pushVett( double* vet, double x ){
    int i;
    for(i=0;i<N;i++)
        vet[i] = x;
}
