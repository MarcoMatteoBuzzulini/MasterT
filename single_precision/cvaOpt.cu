#include "hip/hip_runtime.h"
//
//  MonteCarlo.cu
//  tesi
//
//  Created by Marco Matteo Buzzulini on 27/11/17.
//  Copyright © 2017 Marco Matteo Buzzulini. All rights reserved.
//

#include "MonteCarlo.h"

#define THREADS 128
#define BLOCKS 512
#define PATH 100
#define SIMPB 131072

extern "C" float host_bsCall ( OptionData );
extern "C" void host_cvaEquityOption(CVA *, int);
extern "C" OptionValue dev_cvaEquityOption(CVA *, int, int, int);
extern "C" void printOption( OptionData o);
extern "C" void Chol( float c[N][N], float a[N][N] );
extern "C" void printMultiOpt( MultiOptionData *o);
extern "C" float randMinMax(float min, float max);

void getRandomSigma( float* std );
void getRandomRho( float* rho );
void pushVett( float* vet, float x );

const float defInt = 0.03;
const float recoveryRate = 0.4;
const float S = 100;
const float K = 100;
const float R = 0.05;
const float V = 0.2;
const float T = 1.f;

int main(int argc, const char * argv[]) {
    /*--------------------------- DATA INSTRUCTION -----------------------------------*/
    CVA cva;
    cva.defInt = defInt;
    cva.lgd = (1 - recoveryRate);
    cva.n = PATH;
    cva.dp = (float*)malloc((cva.n+1)*sizeof(float));
    //cva.fp = (float*)malloc((cva.n+1)*sizeof(float));

    // n+1 because it starts from 1
    cva.ee = (OptionValue *)malloc(sizeof(OptionValue)*(cva.n+1));
    float *bs_price = (float*)malloc(sizeof(float)*(cva.n+1));
    
    OptionData opt;
    OptionValue result;
    printf("\nCVA of an European call Option\nIntensita di default %.2f, LGD %.2f\n",cva.defInt,cva.lgd);
    opt.v = V;
    opt.s = S;
    opt.t = T;
    opt.r = R;
    opt.k = K;
    cva.ns = 1;
    cva.option = opt;
    
    hipEvent_t d_start, d_stop;
    int i, SIMS;
    float difference, dt;
    float GPU_timeSpent=0, CPU_timeSpent=0;
    
	//	CUDA Parameters optimized
    printf("Inserisci il numero di simulazioni Monte Carlo(x131.072): ");
    scanf("%d",&SIMS);
    SIMS *= SIMPB;
    printf("\nScenari di Monte Carlo: %d\n",SIMS);
    
    printOption(cva.option);
    bs_price[0] = host_bsCall(cva.option);
    int n = cva.option.t;
    dt = cva.option.t/(float)cva.n;
    for(i=1;i<cva.n+1;i++){
        if((cva.option.t -= dt)<0)
            bs_price[i] = 0;
        else
            bs_price[i] = host_bsCall(cva.option);
        cva.option.t = n;
    }

	// Timer init
    CudaCheck( hipEventCreate( &d_start ));
    CudaCheck( hipEventCreate( &d_stop ));
    /*
    // CPU Monte Carlo
    printf("\nCVA execution on CPU:\n");
    CudaCheck( hipEventRecord( d_start, 0 ));
    host_cvaEquityOption(&cva, SIMS);
    CudaCheck( hipEventRecord( d_stop, 0));
    CudaCheck( hipEventSynchronize( d_stop ));
    CudaCheck( hipEventElapsedTime( &CPU_timeSpent, d_start, d_stop ));
    //CPU_timeSpent /= 1000;
    printf("\nPrezzi Simulati:\n");
    printf("|\ti\t\t|\tPrezzi BS\t| Differenza Prezzi\t|\tPrezzi\t\t|\tDefault Prob\t|\n");
    for(i=0;i<cva.n+1;i++){
        difference = abs(cva.ee[i].Expected - bs_price[i]);
        printf("|\t%f\t|\t%f\t|\t%f\t|\t%f\t|\t%f\t|\n",dt*i,bs_price[i],difference,cva.ee[i].Expected,cva.dp[i]);
    }
    
    printf("\nCVA: %f\n\n",cva.cva);
    printf("\nTotal execution time: %f s\n\n", CPU_timeSpent);
    printf("--------------------------------------------------\n");
     */
    // GPU Monte Carlo
    printf("\nCVA execution on GPU:\n");
    CudaCheck( hipEventRecord( d_start, 0 ));
    result = dev_cvaEquityOption(&cva, BLOCKS, THREADS, SIMS);
    CudaCheck( hipEventRecord( d_stop, 0));
    CudaCheck( hipEventSynchronize( d_stop ));
    CudaCheck( hipEventElapsedTime( &GPU_timeSpent, d_start, d_stop ));
    //GPU_timeSpent /= 1000;

    printf("\nTotal execution time: %f ms\n\n", GPU_timeSpent);
    printf("\nCVA: %f\n\n",result.Expected);
    //printf("Speed up: %f\n\n",CPU_timeSpent/GPU_timeSpent);
   	free(cva.dp);
   	//free(cva.fp);
   	free(cva.ee);
   	free(bs_price);
    return 0;
}

