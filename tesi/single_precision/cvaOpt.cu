#include "hip/hip_runtime.h"
//
//  MonteCarlo.cu
//  tesi
//
//  Created by Marco Matteo Buzzulini on 27/11/17.
//  Copyright © 2017 Marco Matteo Buzzulini. All rights reserved.
//

#include "MonteCarlo.h"

extern "C" float host_bsCall ( OptionData );
extern "C" void host_cvaEquityOption(CVA *, int, int);
extern "C" void dev_cvaEquityOption(CVA *, int , int , int );
extern "C" void printOption( OptionData o);
extern "C" void Chol( float c[N][N], float a[N][N] );
extern "C" void printMultiOpt( MultiOptionData *o);
extern "C" float randMinMax(float min, float max);

void getRandomSigma( float* std );
void getRandomRho( float* rho );
void pushVett( float* vet, float x );

void Parameters(int *numBlocks, int *numThreads);
void memAdjust(hipDeviceProp_t *deviceProp, int *numThreads);
void sizeAdjust(hipDeviceProp_t *deviceProp, int *numBlocks, int *numThreads);

////////////////////////////////////////////////////////////////////////////////////////
//                                      MAIN
////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, const char * argv[]) {
    /*--------------------------- DATA INSTRUCTION -----------------------------------*/
    // Option Data
    if(N>1){
        MultiOptionData option;
        //    Volatility
        option.v[0] = 0.2;
        option.v[1] = 0.3;
        option.v[2] = 0.2;
        //    Spot prices
        option.s[0] = 100;
        option.s[1] = 100;
        option.s[2] = 100;
        //    Weights
        option.w[0] = dw;
        option.w[1] = dw;
        option.w[2] = dw;
        //    Correlations
        option.p[0][0] = 1;
        option.p[0][1] = -0.5;
        option.p[0][2] = -0.5;
        option.p[1][0] = -0.5;
        option.p[1][1] = 1;
        option.p[1][2] = -0.5;
        option.p[2][0] = -0.5;
        option.p[2][1] = -0.5;
        option.p[2][2] = 1;
        //    Drift vectors for the brownians
        option.d[0] = 0;
        option.d[1] = 0;
        option.d[2] = 0;
        
        option.k= 100.f;
        option.r= 0.048790164;
        option.t= 1.f;
    
        if(N!=3){
            srand((unsigned)time(NULL));
            getRandomSigma(option.v);
            getRandomRho(&option.p[0][0]);
            pushVett(option.s,100);
            pushVett(option.w,dw);
            pushVett(option.d,0);
        }
        //    Cholevski factorization
        Chol(cva.option.p, cholRho);
        for(i=0;i<N;i++)
            for(j=0;j<N;j++)
                cva.option.p[i][j]=cholRho[i][j];
    }
    else{
        MultiOptionData option;
        option.v[0] = 0.25;
        option.s[0] = 100;
        option.k= 100.f;
        option.r= 0.05;
        option.t= 1.f;
        option.w[0] = 1;
        option.d[0] = 0;
        option.p[0][0] = 1;
    }
	int numBlocks, numThreads, i, SIMS;
	CVA cva;
		cva.credit.creditspread=150;
		cva.credit.fundingspread=75;
		cva.credit.lgd=60;
		cva.opt = option;
		cva.dp = (float*)malloc((cva.n+1)*sizeof(float));
		cva.fp = (float*)malloc((cva.n+1)*sizeof(float));
		// Puntatore al vettore di prezzi simulati, n+1 perché il primo prezzo è quello originale
		cva.ee = (OptionValue *)malloc(sizeof(OptionValue)*(cva.n+1));
	//float CPU_timeSpent=0, speedup;
    float GPU_timeSpent=0, CPU_timeSpent=0;
    float difference, dt, cholRho[N][N],
    *bs_price = (float*)malloc(sizeof(float)*(cva.n+1));
    hipEvent_t d_start, d_stop;

    printf("Expected Exposures of an Equity Option\n");
	//	Definizione dei parametri CUDA per l'esecuzione in parallelo
    Parameters(&numBlocks, &numThreads);
    printf("Inserisci il numero di simulazioni Monte Carlo(x100.000): ");
    scanf("%d",&SIMS);
    SIMS *= 100000;
    printf("Inserisci il numero di rivalutazioni: ");
    scanf("%d",&cva.n);
    printf("\nScenari di Monte Carlo: %d\n",SIMS);

	//	Print Option details
	printOption(option);

	// Timer init
    CudaCheck( hipEventCreate( &d_start ));
    CudaCheck( hipEventCreate( &d_stop ));

    //	Black & Scholes price
    dt = option.t/(float)cva.n;
    bs_price[0] = host_bsCall(option);
    for(i=1;i<cva.n+1;i++){
    	if((option.t -= dt)<0)
    		bs_price[i] = 0;
    	else
    		bs_price[i] = host_bsCall(option);
    }

    //	Ripristino valore originale del Time to mat
    option.t= 1.f;
    
    // CPU Monte Carlo
    printf("\nCVA execution on CPU:\n");
    CudaCheck( hipEventRecord( d_start, 0 ));
    host_cvaEquityOption(&cva, SIMS);
    CudaCheck( hipEventRecord( d_stop, 0));
    CudaCheck( hipEventSynchronize( d_stop ));
    CudaCheck( hipEventElapsedTime( &CPU_timeSpent, d_start, d_stop ));
    CPU_timeSpent /= 1000;
    printf("\nPrezzi Simulati:\n");
    printf("|\ti\t\t|\tPrezzi BS\t| Differenza Prezzi\t|\tPrezzi\t\t|\tDefault Prob\t|\n");
    for(i=0;i<cva.n+1;i++){
        difference = abs(cva.ee[i].Expected - bs_price[i]);
        printf("|\t%f\t|\t%f\t|\t%f\t|\t%f\t|\t%f\t|\n",dt*i,bs_price[i],difference,cva.ee[i].Expected,cva.dp[i]);
    }
    printf("\nCVA: %f\nFVA: %f\nTotal: %f\n\n",cva.cva,cva.fva,(cva.cva+cva.fva));
    printf("\nTotal execution time: %f s\n\n", CPU_timeSpent);

    // GPU Monte Carlo
    printf("\nCVA execution on GPU:\n");
    CudaCheck( hipEventRecord( d_start, 0 ));
    dev_cvaEquityOption(&cva, numBlocks, numThreads, SIMS);
    CudaCheck( hipEventRecord( d_stop, 0));
    CudaCheck( hipEventSynchronize( d_stop ));
    CudaCheck( hipEventElapsedTime( &GPU_timeSpent, d_start, d_stop ));
    GPU_timeSpent /= 1000;

    printf("\nTotal execution time: %f s\n\n", GPU_timeSpent);

    printf("\nPrezzi Simulati:\n");
   	printf("|\ti\t\t|\tPrezzi BS\t| Differenza Prezzi\t|\tPrezzi\t\t|\tDefault Prob\t|\n");
   	for(i=0;i<cva.n+1;i++){
   		difference = abs(cva.ee[i].Expected - bs_price[i]);
   		printf("|\t%f\t|\t%f\t|\t%f\t|\t%f\t|\t%f\t|\n",dt*i,bs_price[i],difference,cva.ee[i].Expected,cva.dp[i]);
   	}
   	printf("\nCVA: %f\nFVA: %f\nTotal: %f\n\n",cva.cva,cva.fva,(cva.cva+cva.fva));

   	free(cva.dp);
   	free(cva.fp);
   	free(cva.ee);
   	free(bs_price);
    return 0;
}

//Simulation std, rho and covariance matrix
void getRandomSigma( float* std ){
    int i;
    for(i=0;i<N;i++)
        std[i] = randMinMax(0, 1);
}
void getRandomRho( float* rho ){
    int i,j;
    //creating the vectors of rhos
    for(i=0;i<N;i++){
        for(j=i;j<N;j++){
            float r;
            if(i==j)
                r=1;
            else
                r=randMinMax(-1, 1);
            rho[j+i*N] = r;
            rho[i+j*N] = r;
        }
    }
}
void pushVett( float* vet, float x ){
    int i;
    for(i=0;i<N;i++)
        vet[i] = x;
}

///////////////////////////////////
//    ADJUST FUNCTIONS
///////////////////////////////////

void sizeAdjust(hipDeviceProp_t *deviceProp, int *numBlocks, int *numThreads){
    int maxGridSize = deviceProp->maxGridSize[0];
    int maxBlockSize = deviceProp->maxThreadsPerBlock;
    //    Replacing in case of wrong size
    if(*numBlocks > maxGridSize){
        *numBlocks = maxGridSize;
        printf("Warning: maximum size of Grid is %d",*numBlocks);
    }
    if(*numThreads > maxBlockSize){
        *numThreads = maxBlockSize;
        printf("Warning: maximum size of Blocks is %d",*numThreads);
    }
}

void memAdjust(hipDeviceProp_t *deviceProp, int *numThreads){
    size_t maxShared = deviceProp->sharedMemPerBlock;
    size_t maxConstant = deviceProp->totalConstMem;
    int sizeDouble = sizeof(float);
    int numShared = sizeDouble * *numThreads * 2;
    if(sizeof(MultiOptionData) > maxConstant){
        printf("\nWarning: Excess use of constant memory: %zu\n",maxConstant);
        printf("A float variable size is: %d\n",sizeDouble);
        printf("In a MultiOptionData struct there's a consumption of %zu constant memory\n",sizeof(MultiOptionData));
        printf("In this Basket Option there's %d stocks\n",N);
        int maxDim = (int)maxConstant/(sizeDouble*5);
        printf("The optimal number of dims should be: %d stocks\n",maxDim);
    }
    if(numShared > maxShared){
        printf("\nWarning: Excess use of shared memory: %zu\n",maxShared);
        printf("A float variable size is: %d\n",sizeDouble);
        int maxThreads = (int)maxShared / (2*sizeDouble);
        printf("The optimal number of thread should be: %d\n",maxThreads);
    }
    printf("\n");
}

void Parameters(int *numBlocks, int *numThreads){
    hipDeviceProp_t deviceProp;
    CudaCheck(hipGetDeviceProperties(&deviceProp, 0));
    *numThreads = NTHREADS;
    *numBlocks = BLOCKS;
    sizeAdjust(&deviceProp,numBlocks, numThreads);
    memAdjust(&deviceProp, numThreads);
}
