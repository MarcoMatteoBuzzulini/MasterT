#include "hip/hip_runtime.h"
//
//  MonteCarlo.cu
//  tesi
//
//  Created by Marco Matteo Buzzulini on 27/11/17.
//  Copyright © 2017 Marco Matteo Buzzulini. All rights reserved.
//

#include "MonteCarlo.h"

extern "C" float host_bsCall ( OptionData );
extern "C" OptionValue host_vanillaOpt(OptionData, int);
extern "C" OptionValue dev_vanillaOpt(OptionData *, int, int, int);
extern "C" void printOption( OptionData o);

void Parameters(int *numBlocks, int *numThreads);
void memAdjust(hipDeviceProp_t *deviceProp, int *numThreads);
void sizeAdjust(hipDeviceProp_t *deviceProp, int *numBlocks, int *numThreads);

////////////////////////////////////////////////////////////////////////////////////////
//                                      MAIN
////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, const char * argv[]) {
    /*--------------------------- VARIABLES -----------------------------------*/
	// Option Data
	OptionData option;
	option.v = 0.2;
	option.s = 100;
	option.k= 100.f;
	option.r= 0.048790164;
	option.t= 1.f;
	// Simulation
	int numBlocks, numThreads[THREADS], i;
	int SIMS;
	OptionValue CPU_sim, GPU_sim[THREADS];
	float CPU_timeSpent=0, GPU_timeSpent[THREADS], speedup[THREADS];
	float bs_price, difference[THREADS];
	hipEvent_t d_start, d_stop;

    /*--------------------------- START PROGRAM -----------------------------------*/
	printf("Vanilla Option Pricing\n");
	// CUDA parameters for parallel execution
	Parameters(&numBlocks, numThreads);
    printf("Inserisci il numero di simulazioni (x131.072): ");
    scanf("%d",&SIMS);
    SIMS *= 131072;
	//printf("\nScenari di Monte Carlo: %d\n",SIMS);
	//	Print Option details
	printOption(option);
	// Time instructions
    CudaCheck( hipEventCreate( &d_start ));
    CudaCheck( hipEventCreate( &d_stop ));
    //	Black & Scholes price
    bs_price = host_bsCall(option);
    printf("\nPrezzo Black & Scholes: %f\n",bs_price);

    // CPU Monte Carlo
    printf("\nMonte Carlo execution on CPU:\n");
    printf("N^ simulations: %d\n",SIMS);
    CudaCheck( hipEventRecord( d_start, 0 ));
    CPU_sim=host_vanillaOpt(option, SIMS);
    CudaCheck( hipEventRecord( d_stop, 0));
    CudaCheck( hipEventSynchronize( d_stop ));
    CudaCheck( hipEventElapsedTime( &CPU_timeSpent, d_start, d_stop ));
    CPU_timeSpent /= 1000;

    // GPU Monte Carlo
    printf("\nMonte Carlo execution on GPU:\n");
    printf("(NumBlocks, NumSimulations): ( %d ; %d )\n",BLOCKS,SIMS/BLOCKS);
    for(i=0; i<THREADS; i++){
    	CudaCheck( hipEventRecord( d_start, 0 ));
    	GPU_sim[i] = dev_vanillaOpt(&option, numBlocks, numThreads[i],SIMS);
        CudaCheck( hipEventRecord( d_stop, 0));
   	    CudaCheck( hipEventSynchronize( d_stop ));
   	    CudaCheck( hipEventElapsedTime( &GPU_timeSpent[i], d_start, d_stop ));
   	    GPU_timeSpent[i] /= 1000;
   	    difference[i] = abs(GPU_sim[i].Expected - bs_price);
   	    speedup[i] = abs(CPU_timeSpent / GPU_timeSpent[i]);
    }

    // Comparing time spent with the two methods
    printf( "\n-\tResults:\t-\n");
    printf("Simulated price for the option with CPU: Expected price, I.C., time\n%f \n%f \n%f \n",  CPU_sim.Expected, CPU_sim.Confidence, CPU_timeSpent);
    printf("Simulated price for the option with GPU:\n");
    printf("  : NumThreads : Price : Confidence Interval : Difference from BS price :  Time : Speedup :");
    printf("\n");
    for(i=0; i<THREADS; i++){
    	printf("%d \n",numThreads[i]);
    	printf("%f \n",GPU_sim[i].Expected);
    	printf("%f \n",GPU_sim[i].Confidence);
    	printf("%f \n",difference[i]);
    	printf("%f \n",GPU_timeSpent[i]);
    	printf("%.2f \n",speedup[i]);
    	printf("---\n");
    }
    
    CudaCheck( hipEventDestroy( d_start ));
    CudaCheck( hipEventDestroy( d_stop ));
    return 0;
}
///////////////////////////////////
//    ADJUST FUNCTIONS
///////////////////////////////////

void sizeAdjust(hipDeviceProp_t *deviceProp, int *numBlocks, int *numThreads){
    int maxGridSize = deviceProp->maxGridSize[0];
    int maxBlockSize = deviceProp->maxThreadsPerBlock;
    //    Replacing in case of wrong size
    if(*numBlocks > maxGridSize){
        *numBlocks = maxGridSize;
        printf("Warning: maximum size of Grid is %d",*numBlocks);
    }
    if(*numThreads > maxBlockSize){
        *numThreads = maxBlockSize;
        printf("Warning: maximum size of Blocks is %d",*numThreads);
    }
}

void memAdjust(hipDeviceProp_t *deviceProp, int *numThreads){
    size_t maxShared = deviceProp->sharedMemPerBlock;
    size_t maxConstant = deviceProp->totalConstMem;
    int sizeDouble = sizeof(float);
    int numShared = sizeDouble * *numThreads * 2;
    if(sizeof(MultiOptionData) > maxConstant){
        printf("\nWarning: Excess use of constant memory: %zu\n",maxConstant);
        printf("A float variable size is: %d\n",sizeDouble);
        printf("In a MultiOptionData struct there's a consumption of %zu constant memory\n",sizeof(MultiOptionData));
        printf("In this Basket Option there's %d stocks\n",N);
        int maxDim = (int)maxConstant/(sizeDouble*5);
        printf("The optimal number of dims should be: %d stocks\n",maxDim);
    }
    if(numShared > maxShared){
        printf("\nWarning: Excess use of shared memory: %zu\n",maxShared);
        printf("A float variable size is: %d\n",sizeDouble);
        int maxThreads = (int)maxShared / (2*sizeDouble);
        printf("The optimal number of thread should be: %d\n",maxThreads);
    }
}

void Parameters(int *numBlocks, int *numThreads){
    hipDeviceProp_t deviceProp;
    int i = 0;
    CudaCheck(hipGetDeviceProperties(&deviceProp, 0));
    numThreads[0] = 256;
    numThreads[1] = 1024;
    //numThreads[2] = 512;
    //numThreads[3] = 1024;
    *numBlocks = BLOCKS;
    //printf("\nParametri CUDA:\n");
    for (i=0; i<THREADS; i++) {
        sizeAdjust(&deviceProp,numBlocks, &numThreads[i]);
        memAdjust(&deviceProp, &numThreads[i]);
    }
}
