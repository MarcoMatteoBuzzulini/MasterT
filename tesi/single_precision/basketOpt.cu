#include "hip/hip_runtime.h"
//
//  MonteCarlo.cu
//  tesi
//
//  Created by Marco Matteo Buzzulini on 27/11/17.
//  Copyright © 2017 Marco Matteo Buzzulini. All rights reserved.
//

#include "MonteCarlo.h"

extern "C" OptionValue host_basketOpt(MultiOptionData*, int);
extern "C" OptionValue dev_basketOpt(MultiOptionData *, int, int,int);
extern "C" void Chol( float c[N][N], float a[N][N] );
extern "C" void printMultiOpt( MultiOptionData *o);
extern "C" float randMinMax(float min, float max);

void getRandomSigma( float* std );
void getRandomRho( float* rho );
void pushVett( float* vet, float x );

void Parameters(int *numBlocks, int *numThreads);
void memAdjust(hipDeviceProp_t *deviceProp, int *numThreads);
void sizeAdjust(hipDeviceProp_t *deviceProp, int *numBlocks, int *numThreads);

int main(int argc, const char * argv[]) {
    /*--------------------------- VARIABLES -----------------------------------*/
	float dw = (float)1 / N;

	// Option Data
	MultiOptionData option;
	//	Volatility
	option.v[0] = 0.2;
	option.v[1] = 0.3;
	option.v[2] = 0.2;
	//	Spot prices
	option.s[0] = 100;
	option.s[1] = 100;
	option.s[2] = 100;
	//	Weights
	option.w[0] = dw;
	option.w[1] = dw;
	option.w[2] = dw;
	//	Correlations
	option.p[0][0] = 1;
			option.p[0][1] = -0.5;
					option.p[0][2] = -0.5;
	option.p[1][0] = -0.5;
			option.p[1][1] = 1;
					option.p[1][2] = -0.5;
	option.p[2][0] = -0.5;
			option.p[2][1] = -0.5;
					option.p[2][2] = 1;
	//	Drift vectors for the brownians
	option.d[0] = 0;
	option.d[1] = 0;
	option.d[2] = 0;
	option.k= 100.f;
	option.r= 0.048790164;
	option.t= 1.f;

	if(N!=3){
		srand((unsigned)time(NULL));
		getRandomSigma(option.v);
		getRandomRho(&option.p[0][0]);
		pushVett(option.s,100);
		pushVett(option.w,dw);
		pushVett(option.d,0);
	}

	// Simulation variables
	int numBlocks, numThreads[THREADS], SIMS, i, j;
	OptionValue CPU_sim, GPU_sim[THREADS];
	float CPU_timeSpent=0, GPU_timeSpent[THREADS], speedup[THREADS];
	float cholRho[N][N], difference[THREADS];
	// Timer
	// clock_t h_start, h_stop;
	hipEvent_t d_start, d_stop;

	/*--------------------------- START PROGRAM -----------------------------------*/
	printf("Basket Option Pricing\n");
	//	CUDA parameters for parallel execution
	Parameters(&numBlocks, numThreads);
    printf("Inserisci il numero simulazioni (x131.072): ");
    scanf("%d",&SIMS);
    SIMS *= 131072;
	//SIMS = numBlocks*PATH;
	printf("\nScenari di Monte Carlo: %d\n",SIMS);
	//	Print Option details
	if(N<7)
		printMultiOpt(&option);
	else
		printf("\nBasket Option con %d sottostanti\n",N);
    //	Cholevski factorization
    Chol(option.p, cholRho);
    for(i=0;i<N;i++)
    	for(j=0;j<N;j++)
           	option.p[i][j]=cholRho[i][j];
    // Timer init
    CudaCheck( hipEventCreate( &d_start ));
    CudaCheck( hipEventCreate( &d_stop ));
    /* CPU Monte Carlo */
    printf("\nMonte Carlo execution on CPU:\n");
    //printf("N^ simulations: %d\n",SIMS);
    //h_start = clock();
    CudaCheck( hipEventRecord( d_start, 0 ));
    CPU_sim=host_basketOpt(&option, SIMS);
    //h_stop = clock();
    //CPU_timeSpent = ((float)(h_stop - h_start)) / CLOCKS_PER_SEC;
    CudaCheck( hipEventRecord( d_stop, 0));
    CudaCheck( hipEventSynchronize( d_stop ));
    CudaCheck( hipEventElapsedTime( &CPU_timeSpent, d_start, d_stop ));
    CPU_timeSpent /= 1000;

    // GPU Monte Carlo
    printf("\nMonte Carlo execution on GPU:\n");
    //printf("N^ simulations: %d\n",SIMS);
    for(i=0; i<THREADS; i++){
    	CudaCheck( hipEventRecord( d_start, 0 ));
       	GPU_sim[i] = dev_basketOpt(&option, numBlocks, numThreads[i], SIMS);
        CudaCheck( hipEventRecord( d_stop, 0));
        CudaCheck( hipEventSynchronize( d_stop ));
        CudaCheck( hipEventElapsedTime( &GPU_timeSpent[i], d_start, d_stop ));
        GPU_timeSpent[i] /= 1000;
        difference[i] = abs(GPU_sim[i].Expected - CPU_sim.Expected);
        speedup[i] = abs(CPU_timeSpent / GPU_timeSpent[i]);
    }
    // Comparing time spent with the two methods
    printf( "\n-\tResults:\t-\n");
    printf("Simulated price for the option with CPU: Expected price, I.C., time\n%f \n%f \n%f \n", CPU_sim.Expected, CPU_sim.Confidence, CPU_timeSpent);
    printf("Simulated price for the option with GPU:\n");
    printf("  : NumThreads : Price : Confidence Interval : Difference from BS price :  Time : Speedup :");
    printf("\n");
    for(i=0; i<THREADS; i++){
        printf("%d \n",numThreads[i]);
        printf("%f \n",GPU_sim[i].Expected);
        printf("%f \n",GPU_sim[i].Confidence);
        printf("%f \n",difference[i]);
        printf("%f \n",GPU_timeSpent[i]);
        printf("%.2f \n",speedup[i]);
        printf("---\n");
    }
    CudaCheck( hipEventDestroy( d_start ));
    CudaCheck( hipEventDestroy( d_stop ));
    return 0;
}

//////////////////////////////////////////////////////
//    FUNCTIONS
//////////////////////////////////////////////////////////////

//Simulation std, rho and covariance matrix
void getRandomSigma( float* std ){
    int i;
    for(i=0;i<N;i++){
        if((i%2)==0)
            std[i]=0.3;
        else
            std[i]=0.2;
    }
        //std[i] = randMinMax(0, 1);
}
void getRandomRho( float* rho ){
    int i,j;
    //creating the vectors of rhos
    for(i=0;i<N;i++){
        for(j=i;j<N;j++){
            float r;
            if(i==j)
                r=1;
            else
                if(j%2==0)
                    r = 0.5;
                else
                    r= -0.5;
               // r=randMinMax(-1, 1);
            rho[j+i*N] = r;
            rho[i+j*N] = r;
        }
    }
}
void pushVett( float* vet, float x ){
    int i;
    for(i=0;i<N;i++)
        vet[i] = x;
}
///////////////////////////////////
//    ADJUST FUNCTIONS
///////////////////////////////////

void sizeAdjust(hipDeviceProp_t *deviceProp, int *numBlocks, int *numThreads){
    int maxGridSize = deviceProp->maxGridSize[0];
    int maxBlockSize = deviceProp->maxThreadsPerBlock;
    //    Replacing in case of wrong size
    if(*numBlocks > maxGridSize){
        *numBlocks = maxGridSize;
        printf("Warning: maximum size of Grid is %d",*numBlocks);
    }
    if(*numThreads > maxBlockSize){
        *numThreads = maxBlockSize;
        printf("Warning: maximum size of Blocks is %d",*numThreads);
    }
}

void memAdjust(hipDeviceProp_t *deviceProp, int *numThreads){
    size_t maxShared = deviceProp->sharedMemPerBlock;
    size_t maxConstant = deviceProp->totalConstMem;
    int sizeDouble = sizeof(float);
    int numShared = sizeDouble * *numThreads * 2;
    if(sizeof(MultiOptionData) > maxConstant){
        printf("\nWarning: Excess use of constant memory: %zu\n",maxConstant);
        printf("A float variable size is: %d\n",sizeDouble);
        printf("In a MultiOptionData struct there's a consumption of %zu constant memory\n",sizeof(MultiOptionData));
        printf("In this Basket Option there's %d stocks\n",N);
        int maxDim = (int)maxConstant/(sizeDouble*5);
        printf("The optimal number of dims should be: %d stocks\n",maxDim);
    }
    if(numShared > maxShared){
        printf("\nWarning: Excess use of shared memory: %zu\n",maxShared);
        printf("A float variable size is: %d\n",sizeDouble);
        int maxThreads = (int)maxShared / (2*sizeDouble);
        printf("The optimal number of thread should be: %d\n",maxThreads);
    }
    //printf("\n");
}

void Parameters(int *numBlocks, int *numThreads){
    hipDeviceProp_t deviceProp;
    int i = 0;
    CudaCheck(hipGetDeviceProperties(&deviceProp, 0));
    numThreads[0] = 256;
    numThreads[1] = 1024;
    //numThreads[2] = 512;
    //numThreads[3] = 1024;
    //printf("\nParametri CUDA:\n");
    //printf("Scegli il numero di Blocchi: ");
    //scanf("%d",numBlocks);
    *numBlocks = BLOCKS;
    for (i=0; i<THREADS; i++) {
        sizeAdjust(&deviceProp,numBlocks, &numThreads[i]);
        memAdjust(&deviceProp, &numThreads[i]);
    }
}
