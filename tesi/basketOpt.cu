#include "hip/hip_runtime.h"
//
//  MonteCarlo.cu
//  tesi
//
//  Created by Marco Matteo Buzzulini on 27/11/17.
//  Copyright © 2017 Marco Matteo Buzzulini. All rights reserved.
//

#include "MonteCarlo.h"

extern "C" OptionValue host_basketOpt(MultiOptionData*, int);
extern "C" OptionValue dev_basketOpt(MultiOptionData *, int, int);
extern "C" void Chol( double c[N][N], double a[N][N] );
extern "C" void printMultiOpt( MultiOptionData *o);
extern "C" double randMinMax(double min, double max);
//extern "C" void Parameters(int *numBlocks, int *numThreads);

void getRandomSigma( double* std );
void getRandomRho( double* rho );
void pushVett( double* vet, double x );

void Parameters(int *numBlocks, int *numThreads);
void memAdjust(hipDeviceProp_t *deviceProp, int *numThreads);
void sizeAdjust(hipDeviceProp_t *deviceProp, int *numBlocks, int *numThreads);

int main(int argc, const char * argv[]) {
    /*--------------------------- VARIABLES -----------------------------------*/
	double dw = (double)1 / N;

	// Option Data
	MultiOptionData option;
	//	Volatility
	option.v[0] = 0.2;
	option.v[1] = 0.3;
	option.v[2] = 0.2;
	//	Spot prices
	option.s[0] = 100;
	option.s[1] = 100;
	option.s[2] = 100;
	//	Weights
	option.w[0] = dw;
	option.w[1] = dw;
	option.w[2] = dw;
	//	Correlations
	option.p[0][0] = 1;
			option.p[0][1] = -0.5;
					option.p[0][2] = -0.5;
	option.p[1][0] = -0.5;
			option.p[1][1] = 1;
					option.p[1][2] = -0.5;
	option.p[2][0] = -0.5;
			option.p[2][1] = -0.5;
					option.p[2][2] = 1;
	//	Drift vectors for the brownians
	option.d[0] = 0;
	option.d[1] = 0;
	option.d[2] = 0;
	option.k= 100.f;
	option.r= 0.048790164;
	option.t= 1.f;

	if(N!=3){
		srand((unsigned)time(NULL));
		getRandomSigma(option.v);
		getRandomRho(&option.p[0][0]);
		pushVett(option.s,100);
		pushVett(option.w,dw);
		pushVett(option.d,0);
	}

	// Simulation variables
	int numBlocks, numThreads[THREADS], SIMS, i, j;
	OptionValue CPU_sim, GPU_sim[THREADS];
	float CPU_timeSpent=0, GPU_timeSpent[THREADS], speedup[THREADS];
	double cholRho[N][N], difference[THREADS];
	// Timer
	// clock_t h_start, h_stop;
	hipEvent_t d_start, d_stop;

	/*--------------------------- START PROGRAM -----------------------------------*/
	printf("Basket Option Pricing\n");
	//	CUDA parameters for parallel execution
	Parameters(&numBlocks, numThreads);
	SIMS = numBlocks*PATH;
	printf("\nScenari di Monte Carlo: %d\n",SIMS);
	//	Print Option details
	if(N<7)
		printMultiOpt(&option);
	else
		printf("\nBasket Option con %d sottostanti\n",N);
    //	Cholevski factorization
    Chol(option.p, cholRho);
    for(i=0;i<N;i++)
    	for(j=0;j<N;j++)
           	option.p[i][j]=cholRho[i][j];
    // Timer init
    CudaCheck( hipEventCreate( &d_start ));
    CudaCheck( hipEventCreate( &d_stop ));
    /* CPU Monte Carlo */
    printf("\nMonte Carlo execution on CPU:\nN^ simulations: %d\n",SIMS);
    //h_start = clock();
    CudaCheck( hipEventRecord( d_start, 0 ));
    CPU_sim=host_basketOpt(&option, SIMS);
    //h_stop = clock();
    //CPU_timeSpent = ((float)(h_stop - h_start)) / CLOCKS_PER_SEC;
    CudaCheck( hipEventRecord( d_stop, 0));
    CudaCheck( hipEventSynchronize( d_stop ));
    CudaCheck( hipEventElapsedTime( &CPU_timeSpent, d_start, d_stop ));
    CPU_timeSpent /= 1000;

    // GPU Monte Carlo
    printf("\nMonte Carlo execution on GPU:\nN^ simulations: %d\n",SIMS);
    for(i=0; i<THREADS; i++){
    	CudaCheck( hipEventRecord( d_start, 0 ));
       	GPU_sim[i] = dev_basketOpt(&option, numBlocks, numThreads[i]);
        CudaCheck( hipEventRecord( d_stop, 0));
        CudaCheck( hipEventSynchronize( d_stop ));
        CudaCheck( hipEventElapsedTime( &GPU_timeSpent[i], d_start, d_stop ));
        GPU_timeSpent[i] /= 1000;
        difference[i] = abs(GPU_sim[i].Expected - CPU_sim.Expected);
        speedup[i] = abs(CPU_timeSpent / GPU_timeSpent[i]);
    }
    // Comparing time spent with the two methods
    printf( "\n-\tResults:\t-\n");
    printf( "CPU:\n");
    printf("\tSimulated price: € %f with I.C. %f\n", CPU_sim.Expected, CPU_sim.Confidence);
    printf("\tExecution time: %f s (with device function)\n\n", CPU_timeSpent);
    printf("GPU:\n");
    printf("  : NumThreads : Price : Confidence Interval : Difference from CPU price :  Time : Speedup :");
    printf("\n");
    for(i=0; i<THREADS; i++){
        	printf(": \t %d ",numThreads[i]);
        	printf(" \t %f ",GPU_sim[i].Expected);
        	printf(" \t %f  ",GPU_sim[i].Confidence);
        	printf(" \t %f \t",difference[i]);
        	printf(" \t %f ",GPU_timeSpent[i]);
        	printf(" \t %.2f \t",speedup[i]);
        	printf(":\n");
    }
    CudaCheck( hipEventDestroy( d_start ));
    CudaCheck( hipEventDestroy( d_stop ));
    return 0;
}

//////////////////////////////////////////////////////
//    FUNCTIONS
//////////////////////////////////////////////////////////////

//Simulation std, rho and covariance matrix
void getRandomSigma( double* std ){
    int i;
    for(i=0;i<N;i++)
        std[i] = randMinMax(0, 1);
}
void getRandomRho( double* rho ){
    int i,j;
    //creating the vectors of rhos
    for(i=0;i<N;i++){
        for(j=i;j<N;j++){
            double r;
            if(i==j)
                r=1;
            else
                r=randMinMax(-1, 1);
            rho[j+i*N] = r;
            rho[i+j*N] = r;
        }
    }
}
void pushVett( double* vet, double x ){
    int i;
    for(i=0;i<N;i++)
        vet[i] = x;
}
///////////////////////////////////
//    ADJUST FUNCTIONS
///////////////////////////////////

void sizeAdjust(hipDeviceProp_t *deviceProp, int *numBlocks, int *numThreads){
    int maxGridSize = deviceProp->maxGridSize[0];
    int maxBlockSize = deviceProp->maxThreadsPerBlock;
    //    Replacing in case of wrong size
    if(*numBlocks > maxGridSize){
        *numBlocks = maxGridSize;
        printf("Warning: maximum size of Grid is %d",*numBlocks);
    }
    if(*numThreads > maxBlockSize){
        *numThreads = maxBlockSize;
        printf("Warning: maximum size of Blocks is %d",*numThreads);
    }
}

void memAdjust(hipDeviceProp_t *deviceProp, int *numThreads){
    size_t maxShared = deviceProp->sharedMemPerBlock;
    size_t maxConstant = deviceProp->totalConstMem;
    int sizeDouble = sizeof(double);
    int numShared = sizeDouble * *numThreads * 2;
    if(sizeof(MultiOptionData) > maxConstant){
        printf("\nWarning: Excess use of constant memory: %zu\n",maxConstant);
        printf("A double variable size is: %d\n",sizeDouble);
        printf("In a MultiOptionData struct there's a consumption of %zu constant memory\n",sizeof(MultiOptionData));
        printf("In this Basket Option there's %d stocks\n",N);
        int maxDim = (int)maxConstant/(sizeDouble*5);
        printf("The optimal number of dims should be: %d stocks\n",maxDim);
    }
    if(numShared > maxShared){
        printf("\nWarning: Excess use of shared memory: %zu\n",maxShared);
        printf("A double variable size is: %d\n",sizeDouble);
        int maxThreads = (int)maxShared / (2*sizeDouble);
        printf("The optimal number of thread should be: %d\n",maxThreads);
    }
    printf("\n");
}

void Parameters(int *numBlocks, int *numThreads){
    hipDeviceProp_t deviceProp;
    int i = 0;
    CudaCheck(hipGetDeviceProperties(&deviceProp, 0));
    numThreads[0] = 128;
    numThreads[1] = 256;
    numThreads[2] = 512;
    numThreads[3] = 1024;
    printf("\nParametri CUDA:\n");
    printf("Scegli il numero di Blocchi: ");
    scanf("%d",numBlocks);
    for (i=0; i<THREADS; i++) {
        sizeAdjust(&deviceProp,numBlocks, &numThreads[i]);
        memAdjust(&deviceProp, &numThreads[i]);
    }
}
