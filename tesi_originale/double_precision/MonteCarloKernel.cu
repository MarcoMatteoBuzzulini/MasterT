#include "hip/hip_runtime.h"
/*
 * MonteCarloKernel.cu
 *
 *  Created on: 06/feb/2018
 *  Author: marco
 */

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "MonteCarlo.h"

// Struct for Monte Carlo methods
typedef struct{
	OptionValue *h_CallValue, *d_CallValue;
	OptionValue callValue;
    OptionData sopt;
	MultiOptionData mopt;
    hiprandState *RNG;
    int numBlocks, numThreads, numOpt, path;
} dev_MonteCarloData;

/*
 * Error handling from Cuda programming - shane cook
 */
void cuda_error_check(const char * prefix, const char * postfix){
	if (hipPeekAtLastError() != hipSuccess){
		printf("\n%s%s%s", prefix, hipGetErrorString(hipGetLastError()), postfix);
		hipDeviceReset();
		//wait_exit();
		exit(1);
	}
}

// Inizializzazione per Monte Carlo da svolgere una volta sola
void MonteCarlo_init(dev_MonteCarloData *data);
// Liberazione della memoria da svolgere una volta sola
void MonteCarlo_free(dev_MonteCarloData *data);
// Metodo Monte Carlo
void MonteCarlo(dev_MonteCarloData *data);

__device__ __constant__ MultiOptionData MOPTION;
__device__ __constant__ OptionData OPTION;
__device__ __constant__ int N_OPTION, N_PATH;

__device__ void brownianVect(double *bt, hiprandState threadState){
	int i,j;
	double g[N];
	for(i=0;i<N_OPTION;i++)
		g[i]=hiprand_normal(&threadState);
	for(i=0;i<N_OPTION;i++){
		double somma = 0;
		for(j=0;j<N_OPTION;j++)
			somma += MOPTION.p[i][j] * g[j];
		bt[i] = somma;
	}
	for(i=0;i<N_OPTION;i++)
		bt[i] += MOPTION.d[i];
}

__device__ double blackScholes(double *bt){
	int j;
	double s[N], st_sum=0, price;
    for(j=0;j<N_OPTION;j++){
        double geomBt = (MOPTION.r - 0.5 * MOPTION.v[j] * MOPTION.v[j])*MOPTION.t + MOPTION.v[j] * bt[j] * sqrt(MOPTION.t);
	     s[j] = MOPTION.s[j] * exp(geomBt);
    }
	// Third step: Mean price
	for(j=0;j<N_OPTION;j++)
		st_sum += s[j] * MOPTION.w[j];
	// Fourth step: Option payoff
	price = st_sum - MOPTION.k;

    return (price>0)?(price):(0);
}

__global__ void basketOptMonteCarlo(hiprandState * randseed, OptionValue *d_CallValue){
    int i;
    // Parameters for shared memory
    int sumIndex = threadIdx.x;
    int sum2Index = sumIndex + blockDim.x;
    // Parameter for reduction
    int blockIndex = blockIdx.x;

    /*------------------ SHARED MEMORY DICH ----------------*/
    extern __shared__ double s_Sum[];

    // Global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Copy random number state to local memory
    hiprandState threadState = randseed[tid];

    OptionValue sum = {0, 0};

    for( i=sumIndex; i<N_PATH; i+=blockDim.x){
    	double price=0.0f, bt[N];
    	// Random Number Generation
   		brownianVect(bt,threadState);
   		// Price simulation with the Black&Scholes payoff function
        price=blackScholes(bt);

        sum.Expected += price;
        sum.Confidence += price*price;
    }
    // Copy to the shared memory
    s_Sum[sumIndex] = sum.Expected;
    s_Sum[sum2Index] = sum.Confidence;
    __syncthreads();
    // Reduce shared memory accumulators and write final result to global memory
    int halfblock = blockDim.x/2;
    // Reduction in log2(threadBlocks) steps, so threadBlock must be power of 2
    do{
        if ( sumIndex < halfblock ){
            s_Sum[sumIndex] += s_Sum[sumIndex+halfblock];
            s_Sum[sum2Index] += s_Sum[sum2Index+halfblock];
        }
        __syncthreads();
        halfblock /= 2;
    }while ( halfblock != 0 );
    if (sumIndex == 0){
    		d_CallValue[blockIndex].Expected = s_Sum[sumIndex];
    		d_CallValue[blockIndex].Confidence = s_Sum[sum2Index];
    }
}

__global__ void vanillaOptMonteCarlo(hiprandState * randseed, OptionValue *d_CallValue){
    int i;
    // Parameters for shared memory
    int sumIndex = threadIdx.x;
    int sum2Index = sumIndex + blockDim.x;
    
    /*------------------ SHARED MEMORY DICH ----------------*/
    extern __shared__ double s_Sum[];
    
    // Global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Copy random number state to local memory
    hiprandState threadState = randseed[tid];
    
    OptionValue sum = {0, 0};
    
    for( i=sumIndex; i<N_PATH; i+=blockDim.x){
        double price=0.0f, bt, s, geomBt;
        // Random Number Generation
        bt = hiprand_normal(&threadState) * OPTION.t;
        // Price simulation with the Black&Scholes payoff function
        geomBt = (OPTION.r - 0.5 * OPTION.v * OPTION.v) * OPTION.t + OPTION.v * sqrt(bt);
        s = OPTION.s * exp(geomBt);
        price = s - OPTION.k;
        if(price < 0) price = 0;
        sum.Expected += price;
        sum.Confidence += price*price;
    }
    // Copy to the shared memory
    s_Sum[sumIndex] = sum.Expected;
    s_Sum[sum2Index] = sum.Confidence;
    __syncthreads();
    // Reduce shared memory accumulators and write final result to global memory
    int halfblock = blockDim.x/2;
    // Reduction in log2(threadBlocks) steps, so threadBlock must be power of 2
    do{
        if ( sumIndex < halfblock ){
            s_Sum[sumIndex] += s_Sum[sumIndex+halfblock];
            s_Sum[sum2Index] += s_Sum[sum2Index+halfblock];
        }
        __syncthreads();
        halfblock /= 2;
    }while ( halfblock != 0 );
    if (sumIndex == 0){
        d_CallValue[blockIdx.x].Expected = s_Sum[sumIndex];
        d_CallValue[blockIdx.x].Confidence = s_Sum[sum2Index];
    }
}

__global__ void randomSetup( hiprandState *randSeed ){
    // Global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Each thread block gets different seed, threads within a thread block get different sequence numbers
    hiprand_init(blockIdx.x + gridDim.x, threadIdx.x, 0, &randSeed[tid]);
}

void MonteCarlo_init(dev_MonteCarloData *data){
	hipEvent_t start, stop;
	CudaCheck( hipEventCreate( &start ));
    CudaCheck( hipEventCreate( &stop ));
    float time;

    /*--------------- CONSTANT MEMORY ----------------*/
    if( data->numOpt > 1){
        int n_option = data->numOpt;
        CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(N_OPTION),&n_option,sizeof(int)));
    }
    int n_path = data->path;
    CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(N_PATH),&n_path,sizeof(int)));

	// RANDOM NUMBER GENERATION KERNEL
	//Allocate states for pseudo random number generators
	CudaCheck(hipMalloc((void **) &data->RNG, data->numBlocks * data->numThreads * sizeof(hiprandState)));
	//Setup for the random number sequence
    CudaCheck( hipEventRecord( start, 0 ));
    randomSetup<<<data->numBlocks, data->numThreads>>>(data->RNG);
    cuda_error_check("\Errore nel lancio randomSetup: ","\n");
    CudaCheck( hipEventRecord( stop, 0));
    CudaCheck( hipEventSynchronize( stop ));
    CudaCheck( hipEventElapsedTime( &time, start, stop ));
    printf( "RNG done in %f milliseconds\n", time);

    //	Host Memory Allocation
    CudaCheck(hipHostMalloc(&data->h_CallValue, sizeof(OptionValue)*data->numBlocks));
    //	Device Memory Allocation
    CudaCheck(hipMalloc(&data->d_CallValue, sizeof(OptionValue)*data->numBlocks));

    CudaCheck( hipEventDestroy( start ));
    CudaCheck( hipEventDestroy( stop ));
}

void MonteCarlo_free(dev_MonteCarloData *data){
	//Free memory space
	CudaCheck(hipFree(data->RNG));
    CudaCheck(hipHostFree(data->h_CallValue));
    CudaCheck(hipFree(data->d_CallValue));
}

void MonteCarlo(dev_MonteCarloData *data){
	/*----------------- SHARED MEMORY -------------------*/
	int i, numShared = sizeof(double) * data->numThreads * 2;
    
    /*--------------- CONSTANT MEMORY ----------------*/
    if( data->numOpt == 1){
        CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(OPTION),&data->sopt,sizeof(OptionData)));
        vanillaOptMonteCarlo<<<data->numBlocks, data->numThreads, numShared>>>(data->RNG,(OptionValue *)(data->d_CallValue));
        cuda_error_check("\Errore nel lancio vanillaOptMonteCarlo: ","\n");

    }
    else{
        CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(MOPTION),&data->mopt,sizeof(MultiOptionData)));
        basketOptMonteCarlo<<<data->numBlocks, data->numThreads, numShared>>>(data->RNG,(OptionValue *)(data->d_CallValue));
        cuda_error_check("\Errore nel lancio basketOptMonteCarlo: ","\n");
    }

	//MEMORY CPY: prices per block
	CudaCheck(hipMemcpy(data->h_CallValue, data->d_CallValue, data->numBlocks * sizeof(OptionValue), hipMemcpyDeviceToHost));

	// Closing Monte Carlo
	long double sum=0, sum2=0, price, empstd;
    long int nSim = data->numBlocks * data->path;
    for ( i = 0; i < data->numBlocks; i++ ){
    	sum += data->h_CallValue[i].Expected;
	    sum2 += data->h_CallValue[i].Confidence;
	}
	price = exp(-(data->option.r*data->option.t)) * (sum/(double)nSim);
    empstd = sqrt((double)((double)nSim * sum2 - sum * sum)/((double)nSim * (double)(nSim - 1)));
    data->callValue.Confidence = 1.96 * empstd / (double)sqrt((double)nSim);
    data->callValue.Expected = price;
}

extern "C" OptionValue dev_basketOpt(MultiOptionData *option, int numBlocks, int numThreads, int sims){
	dev_MonteCarloData data;
    data.mopt = *option;
    data.numBlocks = numBlocks;
    data.numThreads = numThreads;
    data.numOpt = N;
    data.path = sims / numBlocks;

    MonteCarlo_init(&data);
    MonteCarlo(&data);
    MonteCarlo_free(&data);

    return data.callValue;
}

extern "C" OptionValue dev_vanillaOpt(OptionData *opt, int numBlocks, int numThreads, int sims){
    dev_MonteCarloData data;
    data.option = option;
    data.sopt = *opt;
    data.numBlocks = numBlocks;
    data.numThreads = numThreads;
    data.numOpt = 1;
    data.path = sims / numBlocks;

    MonteCarlo_init(&data);
    MonteCarlo(&data);
    MonteCarlo_free(&data);

    return data.callValue;
}

extern "C" void dev_cvaEquityOption(CVA *cva, int numBlocks, int numThreads, int sims){
    int i;
    double dt = cva->opt.t / (double)cva->n;

    dev_MonteCarloData data;
    // Option
    if(cva->ns ==1)
        data.sopt = cva->option;
    else
        data.mopt = cva->opt;
    // Kernel parameters
    data.numBlocks = numBlocks;
    data.numThreads = numThreads;
    data.numOpt = cva->ns;
    data.path = sims / numBlocks;

    MonteCarlo_init(&data);

    // Original option price
    MonteCarlo(&data);
    cva->ee[0] = data.callValue;

    // Expected Exposures (ee), Default probabilities (dp,fp)
    double sommaProdotto1=0;
    //double sommaProdotto2=0;
	for( i=1; i < (cva->n+1); i++){
		if((data.option.t -= (dt))<0){
			cva->ee[i].Confidence = 0;
			cva->ee[i].Expected = 0;
		}
		else{
			MonteCarlo(&data);
			cva->ee[i] = data.callValue;
		}
        cva->dp[i] = exp(-(dt*i) * cva->defInt) - exp(-(dt*(i+1)) * cva->defInt);
		//cva->fp[i] = exp(-(dt)*(i-1) * cva->credit.fundingspread / 100 / cva->credit.lgd) - exp(-(dt*i) * cva->credit.fundingspread / 100 / cva->credit.lgd );
        sommaProdotto1 += cva->ee[i].Expected * cva->dp[i];
		//sommaProdotto2 += cva->ee[i].Expected * cva->fp[i];
	}
	// CVA and FVA
	cva->cva = sommaProdotto1 * cva->lgd;
	//cva->fva = -sommaProdotto2*cva->credit.lgd;

	// Closing
	MonteCarlo_free(&data);
}
