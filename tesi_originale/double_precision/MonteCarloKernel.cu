#include "hip/hip_runtime.h"
/*
 * MonteCarloKernel.cu
 *
 *  Created on: 06/feb/2018
 *  Author: marco
 */

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "MonteCarlo.h"
#define max(a,b) \
({ __typeof__ (a) _a = (a); \
__typeof__ (b) _b = (b); \
_a > _b ? _a : _b; })

// Struct for Monte Carlo methods
typedef struct{
	OptionValue *h_CallValue, *d_CallValue;
	OptionValue callValue;
    OptionData sopt;
	MultiOptionData mopt;
    hiprandState *RNG;
    int numBlocks, numThreads, numOpt, path;
} dev_MonteCarloData;

/*
 * Error handling from Cuda programming - shane cook
 */
void cuda_error_check(const char * prefix, const char * postfix){
	if (hipPeekAtLastError() != hipSuccess){
		printf("\n%s%s%s", prefix, hipGetErrorString(hipGetLastError()), postfix);
		hipDeviceReset();
		//wait_exit();
		exit(1);
	}
}

// Inizializzazione per Monte Carlo da svolgere una volta sola
void MonteCarlo_init(dev_MonteCarloData *data);
// Liberazione della memoria da svolgere una volta sola
void MonteCarlo_closing(dev_MonteCarloData *data);
// Metodo Monte Carlo
void MonteCarlo(dev_MonteCarloData *data);

__device__ __constant__ MultiOptionData MOPTION;
__device__ __constant__ OptionData OPTION;
__device__ __constant__ int N_OPTION, N_PATH, N_GRID;
__device__ __constant__ float INTDEF, LGD;

__device__ void brownianVect(double *bt, hiprandState *threadState){
	int i,j;
	double g[N];
	for(i=0;i<N_OPTION;i++)
		g[i]=hiprand_normal(threadState);
	for(i=0;i<N_OPTION;i++){
		double somma = 0;
		for(j=0;j<N_OPTION;j++)
			somma += MOPTION.p[i][j] * g[j];
		bt[i] = somma;
	}
	for(i=0;i<N_OPTION;i++)
		bt[i] += MOPTION.d[i];
}

__device__ double basketPayoff(double *bt){
	int j;
	double s[N], st_sum=0, price;
    for(j=0;j<N_OPTION;j++)
        s[j] = MOPTION.s[j] * exp((MOPTION.r - 0.5 * MOPTION.v[j] * MOPTION.v[j])*MOPTION.t+MOPTION.v[j] * bt[j] * sqrt(MOPTION.t));
	// Third step: Mean price
	for(j=0;j<N_OPTION;j++)
		st_sum += s[j] * MOPTION.w[j];
	// Fourth step: Option payoff
	price = st_sum - MOPTION.k;

    return max(price,0);
}

__device__ double geomBrownian( double *s, double *z ){
    double x = (OPTION.r - 0.5 * OPTION.v * OPTION.v) * OPTION.t + OPTION.v * sqrt(OPTION.t) * *z;
    return *s * exp(x);
}

__device__ double callPayoff(hiprandState *threadState){
    double z = hiprand_normal(threadState);
    double sT = OPTION.s * exp((OPTION.r - 0.5 * OPTION.v * OPTION.v) * OPTION.t + OPTION.v * sqrt(OPTION.t) * z);
    double c = sT - OPTION.k;
    return max( c, 0);
}

__global__ void basketOptMonteCarlo(hiprandState * randseed, OptionValue *d_CallValue){
    int i;
    // Parameters for shared memory
    int sumIndex = threadIdx.x;
    int sum2Index = sumIndex + blockDim.x;

    /*------------------ SHARED MEMORY DICH ----------------*/
    extern __shared__ double s_Sum[];

    // Global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Copy random number state to local memory
    hiprandState threadState = randseed[tid];

    OptionValue sum = {0, 0};

    for( i=sumIndex; i<N_PATH; i+=blockDim.x){
    	double price=0.0f, bt[N];
    	// Random Number Generation
   		brownianVect(bt,&threadState);
   		// Price simulation with the basket call option payoff function
        price=basketPayoff(bt);
        // Mean sum
        sum.Expected += price;
        sum.Confidence += price*price;
    }
    // Copy to the shared memory
    s_Sum[sumIndex] = sum.Expected;
    s_Sum[sum2Index] = sum.Confidence;
    __syncthreads();
    // Reduce shared memory accumulators and write final result to global memory
    int halfblock = blockDim.x/2;
    // Reduction in log2(threadBlocks) steps, so threadBlock must be power of 2
    do{
        if ( sumIndex < halfblock ){
            s_Sum[sumIndex] += s_Sum[sumIndex+halfblock];
            s_Sum[sum2Index] += s_Sum[sum2Index+halfblock];
        }
        __syncthreads();
        halfblock /= 2;
    }while ( halfblock != 0 );
    if (sumIndex == 0){
    		d_CallValue[blockIdx.x].Expected = s_Sum[sumIndex];
    		d_CallValue[blockIdx.x].Confidence = s_Sum[sum2Index];
    }
}

__global__ void vanillaOptMonteCarlo(hiprandState * randseed, OptionValue *d_CallValue){
    int i;
    // Parameters for shared memory
    int sumIndex = threadIdx.x;
    int sum2Index = sumIndex + blockDim.x;
    
    /*------------------ SHARED MEMORY DICH ----------------*/
    extern __shared__ double s_Sum[];
    
    // Global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Copy random number state to local memory
    hiprandState threadState = randseed[tid];
    
    OptionValue sum = {0, 0};
    
    for( i=sumIndex; i<N_PATH; i+=blockDim.x){
        double price=0.0f;
        // Price simulation with the vanilla call option payoff function
        price = callPayoff(&threadState);
        sum.Expected += price;
        sum.Confidence += price*price;
    }
    // Copy to the shared memory
    s_Sum[sumIndex] = sum.Expected;
    s_Sum[sum2Index] = sum.Confidence;
    __syncthreads();
    // Reduce shared memory accumulators and write final result to global memory
    int halfblock = blockDim.x/2;
    // Reduction in log2(threadBlocks) steps, so threadBlock must be power of 2
    do{
        if ( sumIndex < halfblock ){
            s_Sum[sumIndex] += s_Sum[sumIndex+halfblock];
            s_Sum[sum2Index] += s_Sum[sum2Index+halfblock];
        }
        __syncthreads();
        halfblock /= 2;
    }while ( halfblock != 0 );
    if (sumIndex == 0){
        d_CallValue[blockIdx.x].Expected = s_Sum[sumIndex];
        d_CallValue[blockIdx.x].Confidence = s_Sum[sum2Index];
    }
}

// Test di cva con simulazione percorso sottostante
__global__ void cvaCallOptMC(hiprandState * randseed, OptionValue *d_CallValue){
    int i,j,k;
    // Parameters for shared memory
    int sumIndex = threadIdx.x;
    int sum2Index = sumIndex + blockDim.x;
    
    /*------------------ SHARED MEMORY DICH ----------------*/
    extern __shared__ double s_Sum[];
    
    // Global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Copy random number state to local memory
    hiprandState threadState = randseed[tid];
    // Monte Carlo core
    OptionValue sum = {0, 0};
    double dt = OPTION.t / N_GRID;
    for(k=blockIdx.x; k<gridDim.x; k+=gridDim.x){
        for( i=sumIndex; i<10000; i+=blockDim.x){
            double price=0.0f, mean_price = 0.0f;
            double s[2];
            s[0] = OPTION.s;
            for(j=1; j<N_GRID; j++){
                double z = hiprand_normal(&threadState);
                s[1] = geomBrownian(&s[0], &z);
                double ee = max((((s[1] + s[0])/2)-OPTION.k),0);
                double dp = exp(-(dt*j-1) * (double)INTDEF) - exp(-(dt*j) * (double)INTDEF);
                mean_price += ee * dp * exp(-(dt*i) * OPTION.r);
                s[0] = s[1];
            }
            price = mean_price * LGD;
            sum.Expected += price;
            sum.Confidence += price*price;
        }
        // Copy to the shared memory
        s_Sum[sumIndex] = sum.Expected;
        s_Sum[sum2Index] = sum.Confidence;
        __syncthreads();
        // Reduce shared memory accumulators and write final result to global memory
        int halfblock = blockDim.x/2;
        // Reduction in log2(threadBlocks) steps, so threadBlock must be power of 2
        do{
            if ( sumIndex < halfblock ){
                s_Sum[sumIndex] += s_Sum[sumIndex+halfblock];
                s_Sum[sum2Index] += s_Sum[sum2Index+halfblock];
            }
            __syncthreads();
            halfblock /= 2;
        }while ( halfblock != 0 );
    }
    if (sumIndex == 0){
        d_CallValue[blockIdx.x].Expected = s_Sum[sumIndex];
        d_CallValue[blockIdx.x].Confidence = s_Sum[sum2Index];
    }
}

__global__ void randomSetup( hiprandState *randSeed ){
    // Global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Each thread block gets different seed, threads within a thread block get different sequence numbers
    hiprand_init(blockIdx.x + gridDim.x, threadIdx.x, 0, &randSeed[tid]);
}

void MonteCarlo_init(dev_MonteCarloData *data){
	hipEvent_t start, stop;
	CudaCheck( hipEventCreate( &start ));
    CudaCheck( hipEventCreate( &stop ));
    float time;

    /*--------------- CONSTANT MEMORY ----------------*/
    if( data->numOpt > 1){
        int n_option = data->numOpt;
        CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(N_OPTION),&n_option,sizeof(int)));
    }
    
    int n_path = data->path;
    CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(N_PATH),&n_path,sizeof(int)));

	// RANDOM NUMBER GENERATION KERNEL
	//Allocate states for pseudo random number generators
	CudaCheck(hipMalloc((void **) &data->RNG, data->numBlocks * data->numThreads * sizeof(hiprandState)));
	//Setup for the random number sequence
    CudaCheck( hipEventRecord( start, 0 ));
    randomSetup<<<data->numBlocks, data->numThreads>>>(data->RNG);
    cuda_error_check("\Errore nel lancio randomSetup: ","\n");
    CudaCheck( hipEventRecord( stop, 0));
    CudaCheck( hipEventSynchronize( stop ));
    CudaCheck( hipEventElapsedTime( &time, start, stop ));
    printf( "RNG done in %f milliseconds\n", time);

    //	Host Memory Allocation
    CudaCheck(hipHostMalloc(&data->h_CallValue, sizeof(OptionValue)*data->numBlocks));
    //	Device Memory Allocation
    CudaCheck(hipMalloc(&data->d_CallValue, sizeof(OptionValue)*data->numBlocks));

    CudaCheck( hipEventDestroy( start ));
    CudaCheck( hipEventDestroy( stop ));
}

void MonteCarlo_closing(dev_MonteCarloData *data){
	//Free memory space
	CudaCheck(hipFree(data->RNG));
    CudaCheck(hipHostFree(data->h_CallValue));
    CudaCheck(hipFree(data->d_CallValue));
}

void MonteCarlo(dev_MonteCarloData *data){
    double r,t;
	/*----------------- SHARED MEMORY -------------------*/
	int i, numShared = sizeof(double) * data->numThreads * 2;
    
    /*--------------- CONSTANT MEMORY ----------------*/
    if( data->numOpt == 1){
        r = data->sopt.r;
        t = data->sopt.t;
        CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(OPTION),&data->sopt,sizeof(OptionData)));
        vanillaOptMonteCarlo<<<data->numBlocks, data->numThreads, numShared>>>(data->RNG,(OptionValue *)(data->d_CallValue));
        cuda_error_check("\Errore nel lancio vanillaOptMonteCarlo: ","\n");

    }
    else{
        r = data->mopt.r;
        t = data->mopt.t;
        CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(MOPTION),&data->mopt,sizeof(MultiOptionData)));
        basketOptMonteCarlo<<<data->numBlocks, data->numThreads, numShared>>>(data->RNG,(OptionValue *)(data->d_CallValue));
        cuda_error_check("\Errore nel lancio basketOptMonteCarlo: ","\n");
    }

	//MEMORY CPY: prices per block
	CudaCheck(hipMemcpy(data->h_CallValue, data->d_CallValue, data->numBlocks * sizeof(OptionValue), hipMemcpyDeviceToHost));

	// Closing Monte Carlo
	long double sum=0, sum2=0, price, empstd;
    long int nSim = data->numBlocks * data->path;
    for ( i = 0; i < data->numBlocks; i++ ){
    	sum += data->h_CallValue[i].Expected;
	    sum2 += data->h_CallValue[i].Confidence;
	}
	price = exp(-r*t) * (sum/(double)nSim);
    empstd = sqrt((double)((double)nSim * sum2 - sum * sum)/((double)nSim * (double)(nSim - 1)));
    data->callValue.Confidence = 1.96 * empstd / (double)sqrt((double)nSim);
    data->callValue.Expected = price;
}

void cvaMonteCarlo(dev_MonteCarloData *data, float intdef, float lgd){
    /*----------------- SHARED MEMORY -------------------*/
    int i, numShared = sizeof(double) * data->numThreads * 2;
    if( data->numOpt == 1){
         /*--------------- CONSTANT MEMORY ----------------*/
        CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(INTDEF),&intdef,sizeof(float)));
        CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(LGD),&lgd,sizeof(float)));
        CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(OPTION),&data->sopt,sizeof(OptionData)));
        cvaCallOptMC<<<data->numBlocks, data->numThreads, numShared>>>(data->RNG,(OptionValue *)(data->d_CallValue));
        cuda_error_check("\Errore nel lancio cvaCallOptMC: ","\n");
    }
    //MEMORY CPY: prices per block
    CudaCheck(hipMemcpy(data->h_CallValue, data->d_CallValue, data->numBlocks * sizeof(OptionValue), hipMemcpyDeviceToHost));
    // Closing Monte Carlo
    long double sum=0, sum2=0, price, empstd;
    long int nSim = data->numBlocks * data->path;
    for ( i = 0; i < data->numBlocks; i++ ){
        sum += data->h_CallValue[i].Expected;
        sum2 += data->h_CallValue[i].Confidence;
    }
    price = sum/(double)nSim;
    empstd = sqrt((double)((double)nSim * sum2 - sum * sum)/((double)nSim * (double)(nSim - 1)));
    data->callValue.Confidence = 1.96 * empstd / (double)sqrt((double)nSim);
    data->callValue.Expected = price;
}

extern "C" OptionValue dev_basketOpt(MultiOptionData *option, int numBlocks, int numThreads, int sims){
	dev_MonteCarloData data;
    data.mopt = *option;
    data.numBlocks = numBlocks;
    data.numThreads = numThreads;
    data.numOpt = N;
    data.path = sims / numBlocks;

    MonteCarlo_init(&data);
    MonteCarlo(&data);
    MonteCarlo_closing(&data);
    
    return data.callValue;
}

extern "C" OptionValue dev_vanillaOpt(OptionData *opt, int numBlocks, int numThreads, int sims){
    dev_MonteCarloData data;
    data.sopt = *opt;
    data.numBlocks = numBlocks;
    data.numThreads = numThreads;
    data.numOpt = 1;
    data.path = sims / numBlocks;

    MonteCarlo_init(&data);
    MonteCarlo(&data);
    MonteCarlo_closing(&data);

    return data.callValue;
}

extern "C" void dev_cvaEquityOption(CVA *cva, int numBlocks, int numThreads, int sims){
    int i;
    double dt, t;
    dev_MonteCarloData data;
    // Option
    if(cva->ns ==1){
        data.sopt = cva->option;
        dt = cva->option.t / (double)cva->n;
        t = cva->option.t;
    }
    else{
        data.mopt = cva->opt;
        dt = cva->opt.t / (double)cva->n;
        t = cva->opt.t;
    }
    // Kernel parameters
    data.numBlocks = numBlocks;
    data.numThreads = numThreads;
    data.numOpt = cva->ns;
    data.path = sims / numBlocks;

    MonteCarlo_init(&data);

    // Original option price
    MonteCarlo(&data);
    cva->ee[0] = data.callValue;

    // Expected Exposures (ee), Default probabilities (dp,fp)
    double sommaProdotto1=0;
    //double sommaProdotto2=0;
	for( i=1; i < (cva->n+1); i++){
		if((t -= (dt))<0){
			cva->ee[i].Confidence = 0;
			cva->ee[i].Expected = 0;
		}
		else{
            if(cva->ns ==1)
                data.sopt.t = t;
            else
                data.mopt.t = t;
			MonteCarlo(&data);
            //data.callValue.Expected = (data.callValue.Expected + cva->ee[i-1].Expected)/2;
			cva->ee[i] = data.callValue;
		}
        cva->dp[i] = exp(-(dt*i) * cva->defInt) - exp(-(dt*(i+1)) * cva->defInt);
		//cva->fp[i] = exp(-(dt)*(i-1) * cva->credit.fundingspread / 100 / cva->credit.lgd) - exp(-(dt*i) * cva->credit.fundingspread / 100 / cva->credit.lgd );
        sommaProdotto1 += cva->ee[i].Expected * cva->dp[i];
		//sommaProdotto2 += cva->ee[i].Expected * cva->fp[i];
	}
	// CVA and FVA
	cva->cva = sommaProdotto1 * cva->lgd;
	//cva->fva = -sommaProdotto2*cva->credit.lgd;

	// Closing
	MonteCarlo_closing(&data);
}

// Test cva con simulazione percorso sottostante
extern "C" OptionValue dev_cvaEquityOption_opt(CVA *cva, int numBlocks, int numThreads, int sims){
    dev_MonteCarloData data;
    // Option
    if(cva->ns ==1){
        data.sopt = cva->option;
    }
    else{
        data.mopt = cva->opt;
    }
    // Kernel parameters
    data.numBlocks = numBlocks;
    data.numThreads = numThreads;
    data.numOpt = cva->ns;
    data.path = sims / numBlocks;
    
    MonteCarlo_init(&data);
    cvaMonteCarlo(&data, (float)cva->defInt, (float)cva->lgd);
    
    // Closing
    MonteCarlo_closing(&data);
    
    return data.callValue;
}


