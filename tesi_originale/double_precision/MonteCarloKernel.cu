#include "hip/hip_runtime.h"
/*
 * MonteCarloKernel.cu
 *
 *  Created on: 06/feb/2018
 *  Author: marco
 */

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "MonteCarlo.h"

// Struct for Monte Carlo methods
typedef struct{
	OptionValue *h_CallValue, *d_CallValue;
	OptionValue callValue;
	MultiOptionData option;
    hiprandState *RNG;
    int numBlocks, numThreads, numOpt, path;
} dev_MonteCarloData;

/*
 * Error handling from Cuda programming - shane cook
 */
void cuda_error_check(const char * prefix, const char * postfix){
	if (hipPeekAtLastError() != hipSuccess){
		printf("\n%s%s%s", prefix, hipGetErrorString(hipGetLastError()), postfix);
		hipDeviceReset();
		//wait_exit();
		exit(1);
	}
}

// Inizializzazione per Monte Carlo da svolgere una volta sola
void MonteCarlo_init(dev_MonteCarloData *data);
// Liberazione della memoria da svolgere una volta sola
void MonteCarlo_free(dev_MonteCarloData *data);
// Metodo Monte Carlo
void MonteCarlo(dev_MonteCarloData *data);

__device__ __constant__ MultiOptionData OPTION;
__device__ __constant__ int N_OPTION, N_PATH;

__device__ void brownianVect(double *bt, hiprandState threadState){
	int i,j;
	double g[N];
	for(i=0;i<N_OPTION;i++)
		g[i]=hiprand_normal(&threadState);
	for(i=0;i<N_OPTION;i++){
		double somma = 0;
		for(j=0;j<N_OPTION;j++)
			somma += OPTION.p[i][j] * g[j];
		bt[i] = somma;
	}
	for(i=0;i<N_OPTION;i++)
		bt[i] += OPTION.d[i];
}

__device__ double blackScholes(double *bt){
	int j;
	double s[N], st_sum=0, price;
    for(j=0;j<N_OPTION;j++){
        double geomBt = (OPTION.r - 0.5 * OPTION.v[j] * OPTION.v[j])*OPTION.t + OPTION.v[j] * bt[j] * sqrt(OPTION.t);
	     s[j] = OPTION.s[j] * exp(geomBt);
    }
	// Third step: Mean price
	for(j=0;j<N_OPTION;j++)
		st_sum += s[j] * OPTION.w[j];
	// Fourth step: Option payoff
	price = st_sum - OPTION.k;

    return (price>0)?(price):(0);
}

__global__ void MultiMCBasketOptKernel(hiprandState * randseed, OptionValue *d_CallValue){
    int i;
    // Parameters for shared memory
    int sumIndex = threadIdx.x;
    int sum2Index = sumIndex + blockDim.x;
    // Parameter for reduction
    int blockIndex = blockIdx.x;

    /*------------------ SHARED MEMORY DICH ----------------*/
    extern __shared__ double s_Sum[];

    // Global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Copy random number state to local memory
    hiprandState threadState = randseed[tid];

    OptionValue sum = {0, 0};

    for( i=sumIndex; i<N_PATH; i+=blockDim.x){
    	double price=0.0f, bt[N];
    	// Random Number Generation
   		brownianVect(bt,threadState);
   		// Price simulation with the Black&Scholes payoff function
        price=blackScholes(bt);

        sum.Expected += price;
        sum.Confidence += price*price;
    }
    // Copy to the shared memory
    s_Sum[sumIndex] = sum.Expected;
    s_Sum[sum2Index] = sum.Confidence;
    __syncthreads();
    // Reduce shared memory accumulators and write final result to global memory
    int halfblock = blockDim.x/2;
    // Reduction in log2(threadBlocks) steps, so threadBlock must be power of 2
    do{
        if ( sumIndex < halfblock ){
            s_Sum[sumIndex] += s_Sum[sumIndex+halfblock];
            s_Sum[sum2Index] += s_Sum[sum2Index+halfblock];
        }
        __syncthreads();
        halfblock /= 2;
    }while ( halfblock != 0 );
    if (sumIndex == 0){
    		d_CallValue[blockIndex].Expected = s_Sum[sumIndex];
    		d_CallValue[blockIndex].Confidence = s_Sum[sum2Index];
    }
}

__global__ void randomSetup( hiprandState *randSeed ){
    // Global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Each thread block gets different seed, threads within a thread block get different sequence numbers
    hiprand_init(blockIdx.x + gridDim.x, threadIdx.x, 0, &randSeed[tid]);
}

void MonteCarlo_init(dev_MonteCarloData *data){
	hipEvent_t start, stop;
	CudaCheck( hipEventCreate( &start ));
    CudaCheck( hipEventCreate( &stop ));
    float time;

    int n_option = data->numOpt;
    int n_path = data->path;

    /*--------------- CONSTANT MEMORY ----------------*/
    CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(N_OPTION),&n_option,sizeof(int)));
    CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(N_PATH),&n_path,sizeof(int)));

	// RANDOM NUMBER GENERATION KERNEL
	//Allocate states for pseudo random number generators
	CudaCheck(hipMalloc((void **) &data->RNG, data->numBlocks * data->numThreads * sizeof(hiprandState)));
	//Setup for the random number sequence
    CudaCheck( hipEventRecord( start, 0 ));
    randomSetup<<<data->numBlocks, data->numThreads>>>(data->RNG);
    cuda_error_check("\Errore nel lancio randomSetup: ","\n");
    CudaCheck( hipEventRecord( stop, 0));
    CudaCheck( hipEventSynchronize( stop ));
    CudaCheck( hipEventElapsedTime( &time, start, stop ));
    printf( "RNG done in %f milliseconds\n", time);

    //	Host Memory Allocation
    CudaCheck(hipHostMalloc(&data->h_CallValue, sizeof(OptionValue)*data->numBlocks));
    //	Device Memory Allocation
    CudaCheck(hipMalloc(&data->d_CallValue, sizeof(OptionValue)*data->numBlocks));

    CudaCheck( hipEventDestroy( start ));
    CudaCheck( hipEventDestroy( stop ));
}

void MonteCarlo_free(dev_MonteCarloData *data){
	//Free memory space
	CudaCheck(hipFree(data->RNG));
    CudaCheck(hipHostFree(data->h_CallValue));
    CudaCheck(hipFree(data->d_CallValue));
}

void MonteCarlo(dev_MonteCarloData *data){
	/*--------------- CONSTANT MEMORY ----------------*/
	CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(OPTION),&data->option,sizeof(MultiOptionData)));

	/*----------------- SHARED MEMORY -------------------*/
	int i, numShared = sizeof(double) * data->numThreads * 2;

	MultiMCBasketOptKernel<<<data->numBlocks, data->numThreads, numShared>>>(data->RNG,(OptionValue *)(data->d_CallValue));
	cuda_error_check("\Errore nel lancio MultiMCBasketOptKernel: ","\n");

	//MEMORY CPY: prices per block
	CudaCheck(hipMemcpy(data->h_CallValue, data->d_CallValue, data->numBlocks * sizeof(OptionValue), hipMemcpyDeviceToHost));

	// Closing Monte Carlo
	long double sum=0, sum2=0, price, empstd;
    long int nSim = data->numBlocks * data->path;
    for ( i = 0; i < data->numBlocks; i++ ){
    	sum += data->h_CallValue[i].Expected;
	    sum2 += data->h_CallValue[i].Confidence;
	}
	price = exp(-(data->option.r*data->option.t)) * (sum/(double)nSim);
    empstd = sqrt((double)((double)nSim * sum2 - sum * sum)/((double)nSim * (double)(nSim - 1)));
    data->callValue.Confidence = 1.96 * empstd / (double)sqrt((double)nSim);
    data->callValue.Expected = price;
}

extern "C" OptionValue dev_basketOpt(MultiOptionData *option, int numBlocks, int numThreads, int sims){
	dev_MonteCarloData data;
	    data.option = *option;
	    data.numBlocks = numBlocks;
	    data.numThreads = numThreads;
	    data.numOpt = N;
	    data.path = sims / numBlocks;

    MonteCarlo_init(&data);
    MonteCarlo(&data);
    MonteCarlo_free(&data);

    return data.callValue;
}

extern "C" OptionValue dev_vanillaOpt(OptionData *opt, int numBlocks, int numThreads, int sims){
	MultiOptionData option;
		option.w[0] = 1;
		option.d[0] = 0;
		option.p[0][0] = 1;
		option.s[0] = opt->s;
		option.v[0] = opt->v;
		option.k = opt->k;
		option.r = opt->r;
		option.t = opt->t;

    dev_MonteCarloData data;
    	data.option = option;
    	data.numBlocks = numBlocks;
    	data.numThreads = numThreads;
    	data.numOpt = 1;
    	data.path = sims / numBlocks;

    MonteCarlo_init(&data);
    MonteCarlo(&data);
    MonteCarlo_free(&data);

    return data.callValue;
}

extern "C" void dev_cvaEquityOption(CVA *cva, int numBlocks, int numThreads, int sims){
    int i;
    double dt = cva->opt.t / (double)cva->n;

    dev_MonteCarloData data;
    // Option
    data.option = cva->opt;
    // Kernel parameters
    data.numBlocks = numBlocks;
    data.numThreads = numThreads;
    data.numOpt = N;
    data.path = sims / numBlocks;

    MonteCarlo_init(&data);

    // Original option price
    MonteCarlo(&data);
    cva->ee[0] = data.callValue;

    // Expected Exposures (ee), Default probabilities (dp,fp)
    double sommaProdotto1=0;
    //double sommaProdotto2=0;
	for( i=1; i < (cva->n+1); i++){
		if((data.option.t -= (dt))<0){
			cva->ee[i].Confidence = 0;
			cva->ee[i].Expected = 0;
		}
		else{
			MonteCarlo(&data);
			cva->ee[i] = data.callValue;
		}
        cva->dp[i] = exp(-(dt*i) * cva->defInt) - exp(-(dt*(i+1)) * cva->defInt);
		//cva->fp[i] = exp(-(dt)*(i-1) * cva->credit.fundingspread / 100 / cva->credit.lgd) - exp(-(dt*i) * cva->credit.fundingspread / 100 / cva->credit.lgd );
        sommaProdotto1 += cva->ee[i].Expected * cva->dp[i];
		//sommaProdotto2 += cva->ee[i].Expected * cva->fp[i];
	}
	// CVA and FVA
	cva->cva = sommaProdotto1 * cva->lgd;
	//cva->fva = -sommaProdotto2*cva->credit.lgd;

	// Closing
	MonteCarlo_free(&data);
}
