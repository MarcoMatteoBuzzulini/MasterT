#include "hip/hip_runtime.h"
//
//  MonteCarlo.cu
//  tesi
//
//  Created by Marco Matteo Buzzulini on 27/11/17.
//  Copyright © 2017 Marco Matteo Buzzulini. All rights reserved.
//

#include "MonteCarlo.h"

#define NTHREADS 2
#define THREADS 256
#define BLOCKS 512
#define SIMPB 131072

extern "C" double host_bsCall ( OptionData );
extern "C" OptionValue host_vanillaOpt(OptionData, int);
extern "C" OptionValue dev_vanillaOpt(OptionData *, int, int, int);
extern "C" void printOption( OptionData o);

const double S = 100;
const double K = 100;
const double R = 0.05;
const double V = 0.2;
const double T = 1.f;

int main(int argc, const char * argv[]) {
    /*------------------------- VARIABLES ------------------------------*/
	// Option Data
	OptionData option;
	option.v = V;
	option.s = S;
	option.k= K;
	option.r= R;
	option.t= T;
	// Simulation
	int numBlocks, numThreads[NTHREADS], i;
	int SIMS;
	OptionValue CPU_sim, GPU_sim[NTHREADS];
	float CPU_timeSpent=0, GPU_timeSpent[NTHREADS], speedup[NTHREADS];
	double bs_price, difference[NTHREADS];
	hipEvent_t d_start, d_stop;

    /*----------------------- START PROGRAM ----------------------------*/
	printf("Vanilla Option Pricing\n");
	// CUDA parameters for parallel execution
    numBlocks = BLOCKS;
    numThreads[0] = THREADS;
    numThreads[1] = 1024;
    printf("Inserisci il numero di simulazioni (x131.072): ");
    scanf("%d",&SIMS);
    SIMS *= SIMPB;
	printf("\nScenari di Monte Carlo: %d\n",SIMS);
	//	Print Option details
	printOption(option);
	// Time instructions
    CudaCheck( hipEventCreate( &d_start ));
    CudaCheck( hipEventCreate( &d_stop ));
    //	Black & Scholes price
    bs_price = host_bsCall(option);
    printf("\nPrezzo Black & Scholes: %f\n",bs_price);

    // CPU Monte Carlo
    printf("\nMonte Carlo execution on CPU:\n");
    printf("N^ simulations: %d\n",SIMS);
    CudaCheck( hipEventRecord( d_start, 0 ));
    CPU_sim=host_vanillaOpt(option, SIMS);
    CudaCheck( hipEventRecord( d_stop, 0));
    CudaCheck( hipEventSynchronize( d_stop ));
    CudaCheck( hipEventElapsedTime( &CPU_timeSpent, d_start, d_stop ));
    CPU_timeSpent /= 1000;

    // GPU Monte Carlo
    printf("\nMonte Carlo execution on GPU:\n");
    printf("(NumBlocks, NumSimulations): ( %d ; %d )\n",BLOCKS,SIMS/BLOCKS);
    for(i=0; i<NTHREADS; i++){
    	CudaCheck( hipEventRecord( d_start, 0 ));
    	GPU_sim[i] = dev_vanillaOpt(&option, numBlocks, numThreads[i],SIMS);
        CudaCheck( hipEventRecord( d_stop, 0));
   	    CudaCheck( hipEventSynchronize( d_stop ));
   	    CudaCheck( hipEventElapsedTime( &GPU_timeSpent[i], d_start, d_stop ));
   	    GPU_timeSpent[i] /= 1000;
   	    difference[i] = abs(GPU_sim[i].Expected - bs_price);
   	    speedup[i] = abs(CPU_timeSpent / GPU_timeSpent[i]);
    }

    // Comparing time spent with the two methods
    printf( "\n-\tResults:\t-\n");
    printf("Simulated price for the option with CPU: Expected price, I.C., time\n%f \n%f \n%f \n",  CPU_sim.Expected, CPU_sim.Confidence, CPU_timeSpent);
    printf("Simulated price for the option with GPU:\n");
    printf("  : NumThreads : Price : Confidence Interval : Difference from BS price :  Time : Speedup :");
    printf("\n");
    for(i=0; i<NTHREADS; i++){
    	printf("%d \n",numThreads[i]);
    	printf("%f \n",GPU_sim[i].Expected);
    	printf("%f \n",GPU_sim[i].Confidence);
    	printf("%f \n",difference[i]);
    	printf("%f \n",GPU_timeSpent[i]);
    	printf("%.2f \n",speedup[i]);
    	printf("---\n");
    }
    
    CudaCheck( hipEventDestroy( d_start ));
    CudaCheck( hipEventDestroy( d_stop ));
    return 0;
}
