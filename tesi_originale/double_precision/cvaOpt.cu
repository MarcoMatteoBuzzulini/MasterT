#include "hip/hip_runtime.h"
//
//  MonteCarlo.cu
//  tesi
//
//  Created by Marco Matteo Buzzulini on 27/11/17.
//  Copyright © 2017 Marco Matteo Buzzulini. All rights reserved.
//

#include "MonteCarlo.h"

extern "C" double host_bsCall ( OptionData );
extern "C" void host_cvaEquityOption(CVA *, int);
extern "C" void dev_cvaEquityOption(CVA *, int , int , int );
extern "C" void printOption( OptionData o);
extern "C" void Chol( double c[N][N], double a[N][N] );
extern "C" void printMultiOpt( MultiOptionData *o);
extern "C" double randMinMax(double min, double max);

void getRandomSigma( double* std );
void getRandomRho( double* rho );
void pushVett( double* vet, double x );

int main(int argc, const char * argv[]) {
    /*--------------------------- DATA INSTRUCTION -----------------------------------*/
    CVA cva;
    cva.defInt = 0.03;
    cva.lgd = (1 - 0.4);
    cva.n = PATH;
    cva.dp = (double*)malloc((cva.n+1)*sizeof(double));
    //cva.fp = (double*)malloc((cva.n+1)*sizeof(double));

    // n+1 because it starts from 1
    cva.ee = (OptionValue *)malloc(sizeof(OptionValue)*(cva.n+1));
    double *bs_price = (double*)malloc(sizeof(double)*(cva.n+1));

    // Option Data
    MultiOptionData opt;
    char risp;
    printf("CVA: %d periodi \nScelta del sottostante:\n(v = opzione call Eu; b = opzione basket con %d sottostanti)\t", PATH, N);
    scanf(" %s",&risp);
    if(risp == 'b'){
        printf("CVA of an European basket Option\n");
        cva.ns = N;
        double dw = (double)1 / N;
        //    Volatility
        opt.v[0] = 0.2;
        opt.v[1] = 0.3;
        opt.v[2] = 0.2;
        //    Spot prices
        opt.s[0] = 100;
        opt.s[1] = 100;
        opt.s[2] = 100;
        //    Weights
        opt.w[0] = dw;
        opt.w[1] = dw;
        opt.w[2] = dw;
        //    Correlations
        opt.p[0][0] = 1;
        opt.p[0][1] = -0.5;
        opt.p[0][2] = -0.5;
        opt.p[1][0] = -0.5;
        opt.p[1][1] = 1;
        opt.p[1][2] = -0.5;
        opt.p[2][0] = -0.5;
        opt.p[2][1] = -0.5;
        opt.p[2][2] = 1;
        //    Drift vectors for the brownians
        opt.d[0] = 0;
        opt.d[1] = 0;
        opt.d[2] = 0;
        
        if(N!=3){
            getRandomSigma(opt.v);
            getRandomRho(&opt.p[0][0]);
            pushVett(opt.s,100);
            pushVett(opt.w,dw);
            pushVett(opt.d,0);
        }
    }
    else{
        printf("CVA of an European Call Option\n");
        opt.v[0] = 0.2;
        opt.s[0] = 100;
        opt.w[0] = 1;
        opt.d[0] = 0;
        opt.p[0][0] = 1;
        cva.ns = 1;
    }
    opt.k= 100.f;
    opt.r= 0.05;
    opt.t= 1.f;
    cva.opt = opt;
	
    hipEvent_t d_start, d_stop;
    int numBlocks, numThreads, i, j, SIMS;
    double difference, dt, cholRho[N][N];
    float GPU_timeSpent=0, CPU_timeSpent=0;
    
	//	CUDA Parameters optimized
    numThreads = NTHREADS;
    numBlocks = BLOCKS;
    printf("Inserisci il numero di simulazioni Monte Carlo(x131.072): ");
    scanf("%d",&SIMS);
    SIMS *= SIMPB;
    printf("\nScenari di Monte Carlo: %d\n",SIMS);
    
    if(risp == 'b'){
        //    Print Option details
        printMultiOpt(&opt);
        //    Cholevski factorization
        Chol(opt.p, cholRho);
        for(i=0;i<N;i++)
            for(j=0;j<N;j++)
                cva.opt.p[i][j]=cholRho[i][j];
    }else{
        OptionData option;
        option.v = opt.v[0];
        option.s = opt.s[0];
        option.k = opt.k;
        option.r = opt.r;
        option.t = opt.t;
        printOption(option);
        bs_price[0] = host_bsCall(option);
        for(i=1;i<cva.n+1;i++){
            if((opt.t -= dt)<0)
                bs_price[i] = 0;
            else
                bs_price[i] = host_bsCall(option);
        }
    }

	// Timer init
    CudaCheck( hipEventCreate( &d_start ));
    CudaCheck( hipEventCreate( &d_stop ));

    //	Black & Scholes price
    dt = opt.t/(double)cva.n;
    

    //	Restore original Time to mat
    opt.t= 1.f;
    
    // CPU Monte Carlo
    printf("\nCVA execution on CPU:\n");
    CudaCheck( hipEventRecord( d_start, 0 ));
    host_cvaEquityOption(&cva, SIMS);
    CudaCheck( hipEventRecord( d_stop, 0));
    CudaCheck( hipEventSynchronize( d_stop ));
    CudaCheck( hipEventElapsedTime( &CPU_timeSpent, d_start, d_stop ));
    CPU_timeSpent /= 1000;
    printf("\nPrezzi Simulati:\n");
    
        printf("|\ti\t\t|\tPrezzi BS\t| Differenza Prezzi\t|\tPrezzi\t\t|\tDefault Prob\t|\n");
        for(i=0;i<cva.n+1;i++){
            difference = abs(cva.ee[i].Expected - bs_price[i]);
            printf("|\t%f\t|\t%f\t|\t%f\t|\t%f\t|\t%f\t|\n",dt*i,bs_price[i],difference,cva.ee[i].Expected,cva.dp[i]);
        }
    
    printf("\nCVA: %f\n\n",cva.cva);
    printf("\nTotal execution time: %f s\n\n", CPU_timeSpent);
    printf("--------------------------------------------------\n");
    // GPU Monte Carlo
    printf("\nCVA execution on GPU:\n");
    CudaCheck( hipEventRecord( d_start, 0 ));
    dev_cvaEquityOption(&cva, numBlocks, numThreads, SIMS);
    CudaCheck( hipEventRecord( d_stop, 0));
    CudaCheck( hipEventSynchronize( d_stop ));
    CudaCheck( hipEventElapsedTime( &GPU_timeSpent, d_start, d_stop ));
    GPU_timeSpent /= 1000;

    printf("\nTotal execution time: %f s\n\n", GPU_timeSpent);

    printf("\nPrezzi Simulati:\n");
   	printf("|\ti\t\t|\tPrezzi BS\t| Differenza Prezzi\t|\tPrezzi\t\t|\tDefault Prob\t|\n");
   	for(i=0;i<cva.n+1;i++){
   		difference = abs(cva.ee[i].Expected - bs_price[i]);
   		printf("|\t%f\t|\t%f\t|\t%f\t|\t%f\t|\t%f\t|\n",dt*i,bs_price[i],difference,cva.ee[i].Expected,cva.dp[i]);
   	}
    printf("\nCVA: %f\n\n",cva.cva);

   	free(cva.dp);
   	//free(cva.fp);
   	free(cva.ee);
   	free(bs_price);
    return 0;
}

//Simulation std, rho and covariance matrix
void getRandomSigma( double* std ){
    int i,j=0;
    for(i=0;i<N;i++){
        if(j==0){
            std[i]=0.3;
            j=1;
        }
        else{
            std[i]=0.2;
            j=0;
        }
    }
}
void getRandomRho( double* rho ){
    int i,j;
    //creating the vectors of rhos
    for(i=0;i<N;i++){
        for(j=i;j<N;j++){
            double r;
            if(i==j)
                r=1;
            else
                if(j%2==0)
                    r = 0.5;
                else
                    r= -0.5;
            rho[j+i*N] = r;
            rho[i+j*N] = r;
        }
    }
}
void pushVett( double* vet, double x ){
    int i;
    for(i=0;i<N;i++)
        vet[i] = x;
}

