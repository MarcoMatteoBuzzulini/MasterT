#include "hip/hip_runtime.h"
/*
 * MonteCarloKernel.cu
 *
 *  Created on: 06/feb/2018
 *  Author: marco
 */

//#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "MonteCarlo.h"

__device__ __constant__ MultiOptionData OPTION;

__global__ void MultiMCBasketOptKernel(hiprandState * randseed, OptionValue *d_CallValue){
    int i,j;
    int sumIndex = threadIdx.x;
    int sum2Index = sumIndex + blockDim.x;
    int blockIndex = blockIdx.x;

    /*------------------ SHARED MEMORY DICH ----------------*/
    extern __shared__ double s_Sum[];

    //Monte Carlo variables
    double st_sum=0.0f, price;

    //vectors of brownian and ST
    double bt[N];
    double s[N];

    // Global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Copy random number state to local memory
    hiprandState threadState = randseed[tid];

    OptionValue sum = {0, 0};

    for( i=sumIndex; i<PATH; i+=blockDim.x){
        st_sum = 0;
        // First step: Brownian motion
        double g[N];
        // RNGs
        for(j=0;j<N;j++)
        	g[j]=hiprand_normal(&threadState);
        //A*G
        double somma;
        int j,k;
        for(j=0;j<N;j++){
        	somma = 0;
         	for(k=0;k<N;k++)
         		//somma += first->data[i][k]*second->data[k][j];
                somma += OPTION.p[j][k] * g[k];
         	//result->data[i][j] = somma;
            bt[j] = somma;
        }
        //X=m+A*G
        for(j=0;j<N;j++)
            bt[j] += OPTION.d[j];

        //	Second step: Price simulation
        for(j=0;j<N;j++){
                s[j] = OPTION.s[j] * exp((OPTION.r - 0.5 * OPTION.v[j] * OPTION.v[j])*OPTION.t+OPTION.v[j] * bt[j] * sqrt(OPTION.t));
        }

        // Third step: Mean price
        for(j=0;j<N;j++)
            st_sum += s[j] * OPTION.w[j];

        //	Fourth step: Option payoff
        price = st_sum - OPTION.k;
        if(price<0)
            price = 0.0f;

        //	Fifth step:	Monte Carlo price sum
        sum.Expected += price;
        sum.Confidence += price*price;
    }
    s_Sum[sumIndex] = sum.Expected;
    s_Sum[sum2Index] = sum.Confidence;
    __syncthreads();
    //Reduce shared memory accumulators and write final result to global memory
    int halfblock = blockDim.x/2;
    do{
        if ( sumIndex < halfblock ){
            s_Sum[sumIndex] += s_Sum[sumIndex+halfblock];
            s_Sum[sum2Index] += s_Sum[sum2Index+halfblock];
            __syncthreads();
        }
        halfblock /= 2;
    }while ( halfblock != 0 );
    __syncthreads();
    //Keeping the first element for each block using one thread
    if (sumIndex == 0){
    	d_CallValue[blockIndex].Expected = s_Sum[sumIndex];
    	d_CallValue[blockIndex].Confidence = s_Sum[sum2Index];
    }
}

__global__ void randomSetup( hiprandState *randSeed ){
    // Global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Each threadblock gets different seed, threads within a threadblock get different sequence numbers
    hiprand_init(blockIdx.x + gridDim.x, threadIdx.x, 0, &randSeed[tid]);
}

extern "C" OptionValue dev_basketOpt(MultiOptionData *option, int numBlocks, int numThreads){
    int i;
    OptionValue callValue;
    /*----------------- HOST MEMORY -------------------*/
    OptionValue *h_CallValue;
    //Allocation pinned host memory for prices
    CudaCheck(hipHostAlloc(&h_CallValue, sizeof(OptionValue)*(numBlocks),hipHostMallocDefault));

    /*--------------- CONSTANT MEMORY ----------------*/
    CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(OPTION),option,sizeof(MultiOptionData)));

    /*----------------- DEVICE MEMORY -------------------*/
    OptionValue *d_CallValue;
    CudaCheck(hipMalloc(&d_CallValue, sizeof(OptionValue)*(numBlocks)));

    /*----------------- SHARED MEMORY -------------------*/
    int numShared = sizeof(double) * numThreads * 2;

    /*------------ RNGs and TIME VARIABLES --------------*/
    hiprandState *RNG;
    hipEvent_t start, stop;
    CudaCheck( hipEventCreate( &start ));
    CudaCheck( hipEventCreate( &stop ));
    float time;

    // RANDOM NUMBER GENERATION KERNEL
    //Allocate states for pseudo random number generators
    CudaCheck(hipMalloc((void **) &RNG, numBlocks * numThreads * sizeof(hiprandState)));
    //Setup for the random number sequence
    CudaCheck( hipEventRecord( start, 0 ));
    randomSetup<<<numBlocks, numThreads>>>(RNG);
    CudaCheck( hipEventRecord( stop, 0));
    CudaCheck( hipEventSynchronize( stop ));
    CudaCheck( hipEventElapsedTime( &time, start, stop ));
    printf( "RNG done in %f milliseconds\n", time);


    //MONTE CARLO KERNEL
    CudaCheck( hipEventRecord( start, 0 ));
    MultiMCBasketOptKernel<<<numBlocks, numThreads, numShared>>>(RNG,(OptionValue *)(d_CallValue));
    CudaCheck( hipEventRecord( stop, 0));
    CudaCheck( hipEventSynchronize( stop ));
    CudaCheck( hipEventElapsedTime( &time, start, stop ));
    printf( "Monte Carlo simulations done in %f milliseconds\n", time);
    CudaCheck( hipEventDestroy( start ));
    CudaCheck( hipEventDestroy( stop ));

    //MEMORY CPY: prices per block
    CudaCheck(hipMemcpy(h_CallValue, d_CallValue, numBlocks * sizeof(OptionValue), hipMemcpyDeviceToHost));

    // Closing Monte Carlo
    long double sum=0, sum2=0, price, empstd;
    long int nSim = numBlocks * PATH;
    for ( i = 0; i < numBlocks; i++ ){
        sum += h_CallValue[i].Expected;
        sum2 += h_CallValue[i].Confidence;
    }
    price = exp(-(option->r*option->t)) * (sum/(double)nSim);
    empstd = sqrt((double)((double)nSim * sum2 - sum * sum)
                         /((double)nSim * (double)(nSim - 1)));
    callValue.Confidence = 1.96 * empstd / (double)sqrt((double)nSim);
    callValue.Expected = price;

    //Free memory space
    CudaCheck(hipFree(RNG));
    CudaCheck(hipHostFree(h_CallValue));
    CudaCheck(hipFree(d_CallValue));

    return callValue;
}

extern "C" OptionValue dev_vanillaOpt(OptionData *opt, int numBlocks, int numThreads){
    int i;
    OptionValue callValue;
    /*----------------- HOST MEMORY -------------------*/
    OptionValue *h_CallValue;
    //Allocation pinned host memory for prices
    CudaCheck(hipHostAlloc(&h_CallValue, sizeof(OptionValue)*(numBlocks),hipHostMallocDefault));

    /*--------------- CONSTANT MEMORY ----------------*/
    MultiOptionData option;
    option.w[0] = 1;
    option.d[0] = 0;
    option.p[0][0] = 1;
    option.s[0] = opt->s;
    option.v[0] = opt->v;
    option.k = opt->k;
    option.r = opt->r;
    option.t = opt->t;

    CudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(OPTION),option,sizeof(MultiOptionData)));

    /*----------------- DEVICE MEMORY -------------------*/
    OptionValue *d_CallValue;
    CudaCheck(hipMalloc(&d_CallValue, sizeof(OptionValue)*(numBlocks)));

    /*----------------- SHARED MEMORY -------------------*/
    int numShared = sizeof(double) * numThreads * 2;

    /*------------ RNGs and TIME VARIABLES --------------*/
    hiprandState *RNG;
    hipEvent_t start, stop;
    CudaCheck( hipEventCreate( &start ));
    CudaCheck( hipEventCreate( &stop ));
    float time;

    // RANDOM NUMBER GENERATION KERNEL
    //Allocate states for pseudo random number generators
    CudaCheck(hipMalloc((void **) &RNG, numBlocks * numThreads * sizeof(hiprandState)));
    //Setup for the random number sequence
    CudaCheck( hipEventRecord( start, 0 ));
    randomSetup<<<numBlocks, numThreads>>>(RNG);
    CudaCheck( hipEventRecord( stop, 0));
    CudaCheck( hipEventSynchronize( stop ));
    CudaCheck( hipEventElapsedTime( &time, start, stop ));
    printf( "RNG done in %f milliseconds\n", time);


    //MONTE CARLO KERNEL
    CudaCheck( hipEventRecord( start, 0 ));
    MultiMCBasketOptKernel<<<numBlocks, numThreads, numShared>>>(RNG,(OptionValue *)(d_CallValue));
    CudaCheck( hipEventRecord( stop, 0));
    CudaCheck( hipEventSynchronize( stop ));
    CudaCheck( hipEventElapsedTime( &time, start, stop ));
    printf( "Monte Carlo simulations done in %f milliseconds\n", time);
    CudaCheck( hipEventDestroy( start ));
    CudaCheck( hipEventDestroy( stop ));

    //MEMORY CPY: prices per block
    CudaCheck(hipMemcpy(h_CallValue, d_CallValue, numBlocks * sizeof(OptionValue), hipMemcpyDeviceToHost));

    // Closing Monte Carlo
    long double sum=0, sum2=0, price, empstd;
    long int nSim = numBlocks * PATH;
    for ( i = 0; i < numBlocks; i++ ){
        sum += h_CallValue[i].Expected;
        sum2 += h_CallValue[i].Confidence;
    }
    price = exp(-(option->r*option->t)) * (sum/(double)nSim);
    empstd = sqrt((double)((double)nSim * sum2 - sum * sum)
                         /((double)nSim * (double)(nSim - 1)));
    callValue.Confidence = 1.96 * empstd / (double)sqrt((double)nSim);
    callValue.Expected = price;

    //Free memory space
    CudaCheck(hipFree(RNG));
    CudaCheck(hipHostFree(h_CallValue));
    CudaCheck(hipFree(d_CallValue));

    return callValue;
}
