#include "hip/hip_runtime.h"
//
//  MonteCarlo.cu
//  tesi
//
//  Created by Marco Matteo Buzzulini on 27/11/17.
//  Copyright © 2017 Marco Matteo Buzzulini. All rights reserved.
//

#include "MonteCarlo.h"
#include <hip/hip_runtime.h>

// includes, project
#include <helper_functions.h> // Helper functions (utilities, parsing, timing)
#include <hip/hip_runtime_api.h>      // helper functions (cuda error checking and initialization)
#include <multithreading.h>

//	Host Black & Scholes
extern "C" double host_bsCall ( OptionData );

//	Host MonteCarlo
extern "C" OptionValue host_vanillaOpt(OptionData, int);

//	Device MonteCarlo
extern "C" OptionValue dev_vanillaOpt(OptionData *, int, int);

///////////////////////////////////
//	PRINT FUNCTIONS
///////////////////////////////////

void printOption( OptionData o){
    printf("\n-\tOption data\t-\n\n");
    printf("Underlying asset price:\t € %.2f\n", o.s);
    printf("Strike price:\t\t € %.2f\n", o.k);
    printf("Risk free interest rate: %.2f\n", o.r);
    printf("Volatility:\t\t %.2f\n", o.v);
    printf("Time to maturity:\t %.2f %s\n", o.t, (o.t>1)?("years"):("year"));
}


///////////////////////////////////
//	ADJUST FUNCTIONS
///////////////////////////////////

void sizeAdjust(hipDeviceProp_t *deviceProp, int *numBlocks, int *numThreads){
	int maxGridSize = deviceProp->maxGridSize[0];
	int maxBlockSize = deviceProp->maxThreadsPerBlock;
	//	Replacing in case of wrong size
	if(*numBlocks > maxGridSize){
		*numBlocks = maxGridSize;
		printf("Warning: maximum size of Grid is %d",*numBlocks);
	}
	if(*numThreads > maxBlockSize){
		*numThreads = maxBlockSize;
		printf("Warning: maximum size of Blocks is %d",*numThreads);
	}
}

void memAdjust(hipDeviceProp_t *deviceProp, int *numThreads){
		size_t maxShared = deviceProp->sharedMemPerBlock;
		size_t maxConstant = deviceProp->totalConstMem;
		int sizeDouble = sizeof(double);
		int numShared = sizeDouble * *numThreads * 2;
		if(sizeof(MultiOptionData) > maxConstant){
			printf("\nWarning: Excess use of constant memory: %zu\n",maxConstant);
			printf("A double variable size is: %d\n",sizeDouble);
			printf("In a MultiOptionData struct there's a consumption of %zu constant memory\n",sizeof(MultiOptionData));
			printf("In this Basket Option there's %d stocks\n",N);
			int maxDim = (int)maxConstant/(sizeDouble*5);
			printf("The optimal number of dims should be: %d stocks\n",maxDim);
		}
		if(numShared > maxShared){
			printf("\nWarning: Excess use of shared memory: %zu\n",maxShared);
			printf("A double variable size is: %d\n",sizeDouble);
			int maxThreads = (int)maxShared / (2*sizeDouble);
			printf("The optimal number of thread should be: %d\n",maxThreads);
		}
		printf("\n");
}

void optimalAdjust(hipDeviceProp_t *deviceProp, int *numBlocks, int *numThreads){
	int multiProcessors = deviceProp->multiProcessorCount;
	int cudaCoresPM = _ConvertSMVer2Cores(deviceProp->major, deviceProp->minor);
	*numBlocks = multiProcessors * 40;
	*numThreads = pow(2,(int)(log(cudaCoresPM)/log(2)));
	sizeAdjust(deviceProp,numBlocks, numThreads);
}

void choseParameters(int *numBlocks, int *numThreads){
		hipDeviceProp_t deviceProp;
		CudaCheck(hipGetDeviceProperties(&deviceProp, 0));
		char risp;
		printf("\nParametri CUDA:\n");
		printf("Scegli il numero di Blocchi: ");
		scanf("%d",numBlocks);
		printf("Scegli il numero di Threads per blocco: ");
		scanf("%d",numThreads);
		printf("Vuoi ottimizzare i parametri inseriti? (Y/N) ");
		scanf("%s",&risp);
		if((risp=='Y')||(risp=='y'))
			optimalAdjust(&deviceProp,numBlocks, numThreads);
		else
			sizeAdjust(&deviceProp,numBlocks, numThreads);
		memAdjust(&deviceProp,numThreads);
}

////////////////////////////////////////////////////////////////////////////////////////
//                                      MAIN
////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, const char * argv[]) {
    /*--------------------------- DATA INSTRUCTION -----------------------------------*/
	OptionData option;
	option.v = 0.2;
	option.s = 100;
	option.k= 100.f;
	option.r= 0.048790164;
	option.t= 1.f;
	printf("Vanilla Option Pricing\n");

	//	Definizione dei parametri CUDA per l'esecuzione in parallelo
	int numBlocks, numThreads;
	choseParameters(&numBlocks, &numThreads);

	printf("Simulazione di ( %d ; %d )\n",numBlocks, numThreads);
	int SIMS = numBlocks*PATH;

	//	Print Option details
	printOption(option);

    /*---------------- CORE COMPUTATIONS ----------------*/

    OptionValue CPU_sim = {0,0}, GPU_sim = {0,0};
    
    float CPU_timeSpent=0, GPU_timeSpent=0, speedup;
    double price, bs_price, difference;
    //clock_t h_start, h_stop;
    hipEvent_t d_start, d_stop;
    CudaCheck( hipEventCreate( &d_start ));
    CudaCheck( hipEventCreate( &d_stop ));

    //	Black & Scholes price
    bs_price = host_bsCall(option);
    printf("\nPrezzo Black & Scholes: %f\n",bs_price);

    /* CPU Monte Carlo
    printf("\nMonte Carlo execution on CPU:\nN^ simulations: %d\n\n",SIMS);
    h_start = clock();
    //CudaCheck( hipEventRecord( d_start, 0 ));
    CPU_sim=host_vanillaOpt(&option, SIMS);
    h_stop = clock();
    CPU_timeSpent = ((float)(h_stop - h_start)) / CLOCKS_PER_SEC;
    //CudaCheck( hipEventRecord( d_stop, 0));
    //CudaCheck( hipEventSynchronize( d_stop ));
    //CudaCheck( hipEventElapsedTime( &CPU_timeSpent, d_start, d_stop ));
    //CPU_timeSpent /= CLOCKS_PER_SEC;
    
    price = CPU_sim.Expected;
    printf("Simulated price for the basket option: € %f with I.C [ %f;%f ]\n", price, price - CPU_sim.Confidence, price + CPU_sim.Confidence);
    printf("Total execution time: %f s\n\n", CPU_timeSpent);
    */
    // GPU Monte Carlo
    printf("\nMonte Carlo execution on GPU:\nN^ simulations: %d\n",SIMS);
    CudaCheck( hipEventRecord( d_start, 0 ));
    GPU_sim = dev_vanillaOpt(&option, numBlocks, numThreads);
    CudaCheck( hipEventRecord( d_stop, 0));
    CudaCheck( hipEventSynchronize( d_stop ));
    CudaCheck( hipEventElapsedTime( &GPU_timeSpent, d_start, d_stop ));
    GPU_timeSpent /= 1000;
    
    price = GPU_sim.Expected;
    printf("Simulated price for the basket option: € %f with I.C [ %f;%f ]\n", price, price-GPU_sim.Confidence, price + GPU_sim.Confidence);
    printf("Total execution time: %f s\n\n", GPU_timeSpent);
    
    // Comparing time spent with the two methods
    printf( "-\tComparing results:\t-\n");
    difference = abs(price - bs_price);
    speedup = abs(CPU_timeSpent / GPU_timeSpent);
    printf( "The GPU's speedup: %.2f \nDifference from Black & Schole price: %.2f\n", speedup, difference);
    return 0;
}
