#include "hip/hip_runtime.h"
//
//  MonteCarlo.cu
//  tesi
//
//  Created by Marco Matteo Buzzulini on 27/11/17.
//  Copyright © 2017 Marco Matteo Buzzulini. All rights reserved.
//

#include "MonteCarlo.h"
#include <hip/hip_runtime.h>

// includes, project
#include <helper_functions.h> // Helper functions (utilities, parsing, timing)
#include <hip/hip_runtime_api.h>      // helper functions (cuda error checking and initialization)
#include <multithreading.h>

extern "C" OptionValue host_basketOpt(MultiOptionData*, int);
extern "C" OptionValue dev_basketOpt(MultiOptionData *, int, int);

///////////////////////////////////
//	PRINT FUNCTIONS
///////////////////////////////////
void printVect( double *mat, int c ){
    int i,j,r=1;
    for(i=0; i<r; i++){
        printf("\n!\t");
        for(j=0; j<c; j++){
            printf("\t%f\t",mat[j+i*c]);
        }
        printf("\t!");
    }
    printf("\n\n");
}

void printOption( OptionData o){
    printf("\n-\tOption data\t-\n\n");
    printf("Underlying asset price:\t € %.2f\n", o.s);
    printf("Strike price:\t\t € %.2f\n", o.k);
    printf("Risk free interest rate: %.2f %%\n", o.r * 100);
    printf("Volatility:\t\t\t %.2f %%\n", o.v * 100);
    printf("Time to maturity:\t\t %.2f %s\n", o.t, (o.t>1)?("years"):("year"));
}

void printMat( double *mat, int r, int c ){
    int i,j;
    for(i=0; i<r; i++){
        printf("\n!\t");
        for(j=0; j<c; j++){
            printf("\t%f\t",mat[j+i*c]);
        }
        printf("\t!");
    }
    printf("\n\n");
}

void printMultiOpt( MultiOptionData *o){
    printf("\n-\tBasket Option data\t-\n\n");
    printf("Number of assets: %d\n",N);
    printf("Underlying assets prices:\n");
    printVect(o->s, N);
    printf("Volatility:\n");
    printVect(o->v, N);
    printf("Weights:");
    printVect(o->w, N);
    printf("Correlation matrix:\n");
    printMat(&o->p[0][0], N, N);
    printf("Strike price:\t € %.2f\n", o->k);
    printf("Risk free interest rate: %.2f \n", o->r);
    printf("Time to maturity:\t %.2f %s\n", o->t, (o->t>1)?("years"):("year"));
}

///////////////////////////////////
//	ADJUST FUNCTIONS
///////////////////////////////////

void sizeAdjust(hipDeviceProp_t *deviceProp, int *numBlocks, int *numThreads){
	int maxGridSize = deviceProp->maxGridSize[0];
	int maxBlockSize = deviceProp->maxThreadsPerBlock;
	//	Replacing in case of wrong size
	if(*numBlocks > maxGridSize){
		*numBlocks = maxGridSize;
		printf("Warning: maximum size of Grid is %d",*numBlocks);
	}
	if(*numThreads > maxBlockSize){
		*numThreads = maxBlockSize;
		printf("Warning: maximum size of Blocks is %d",*numThreads);
	}
}

void memAdjust(hipDeviceProp_t *deviceProp, int *numThreads){
		size_t maxShared = deviceProp->sharedMemPerBlock;
		size_t maxConstant = deviceProp->totalConstMem;
		int sizeDouble = sizeof(double);
		int numShared = sizeDouble * *numThreads * 2;
		if(sizeof(MultiOptionData) > maxConstant){
			printf("\nWarning: Excess use of constant memory: %zu\n",maxConstant);
			printf("A double variable size is: %d\n",sizeDouble);
			printf("In a MultiOptionData struct there's a consumption of %zu constant memory\n",sizeof(MultiOptionData));
			printf("In this Basket Option there's %d stocks\n",N);
			int maxDim = (int)maxConstant/(sizeDouble*5);
			printf("The optimal number of dims should be: %d stocks\n",maxDim);
		}
		if(numShared > maxShared){
			printf("\nWarning: Excess use of shared memory: %zu\n",maxShared);
			printf("A double variable size is: %d\n",sizeDouble);
			int maxThreads = (int)maxShared / (2*sizeDouble);
			printf("The optimal number of thread should be: %d\n",maxThreads);
		}
		printf("\n");
}

void optimalAdjust(hipDeviceProp_t *deviceProp, int *numBlocks, int *numThreads){
	int multiProcessors = deviceProp->multiProcessorCount;
	int cudaCoresPM = _ConvertSMVer2Cores(deviceProp->major, deviceProp->minor);
	*numBlocks = multiProcessors * 40;
	*numThreads = pow(2,(int)(log(cudaCoresPM)/log(2)))*2;
	sizeAdjust(deviceProp,numBlocks, numThreads);
}

void choseParameters(int *numBlocks, int *numThreads){
		hipDeviceProp_t deviceProp;
		CudaCheck(hipGetDeviceProperties(&deviceProp, 0));
		char risp;
		printf("\nParametri CUDA:\n");
		printf("Scegli il numero di Blocchi: ");
		scanf("%d",numBlocks);
		printf("Scegli il numero di Threads per blocco: ");
		scanf("%d",numThreads);
		printf("Vuoi ottimizzare i parametri inseriti? (Y/N) ");
		scanf("%s",&risp);
		if((risp=='Y')||(risp=='y'))
			optimalAdjust(&deviceProp,numBlocks, numThreads);
		else
			sizeAdjust(&deviceProp,numBlocks, numThreads);
		memAdjust(&deviceProp,numThreads);
}

void Parameters(int *numBlocks, int *numThreads){
		hipDeviceProp_t deviceProp;
		CudaCheck(hipGetDeviceProperties(&deviceProp, 0));
		numThreads[0] = 128;
		numThreads[1] = 256;
		numThreads[2] = 512;
		numThreads[3] = 1024;
		printf("\nParametri CUDA:\n");
		printf("Scegli il numero di Blocchi: ");
		scanf("%d",numBlocks);
}

////////////////////////////////////////////////////////////////////////////////////////
//                                      MAIN
////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, const char * argv[]) {
    /*--------------------------- VARIABLES -----------------------------------*/
	double dw = (double)1 / N;

	// Option Data
	MultiOptionData option;
	//	Volatility
	option.v[0] = 0.2;
	option.v[1] = 0.3;
	option.v[2] = 0.2;
	//	Spot prices
	option.s[0] = 100;
	option.s[1] = 100;
	option.s[2] = 100;
	//	Weights
	option.w[0] = dw;
	option.w[1] = dw;
	option.w[2] = dw;
	//	Correlations
	option.p[0][0] = 1;
			option.p[0][1] = -0.5;
					option.p[0][2] = -0.5;
	option.p[1][0] = -0.5;
			option.p[1][1] = 1;
					option.p[1][2] = -0.5;
	option.p[2][0] = -0.5;
			option.p[2][1] = -0.5;
					option.p[2][2] = 1;
	//	Drift vectors for the brownians
	option.d[0] = 0;
	option.d[1] = 0;
	option.d[2] = 0;
	option.k= 100.f;
	option.r= 0.048790164;
	option.t= 1.f;

	// Simulation variables
	int numBlocks, numThreads[THREADS], SIMS, i, j;
	OptionValue CPU_sim, GPU_sim[THREADS];
	float CPU_timeSpent=0, GPU_timeSpent[THREADS], speedup[THREADS];
	double cholRho[N][N], d_price, h_price, difference[THREADS];
	// Timer
	// clock_t h_start, h_stop;
	hipEvent_t d_start, d_stop;

	/*--------------------------- START PROGRAM -----------------------------------*/
	printf("Basket Option Pricing\n");
	//	CUDA parameters for parallel execution
	Parameters(&numBlocks, numThreads);
	printf("Simulazione di ( %d ; %d )\n",numBlocks, numThreads);
	SIMS = numBlocks*PATH;
	//	Print Option details
	printMultiOpt(&option);
    //	Cholevski factorization
    Chol(option.p, cholRho);
    for(i=0;i<N;i++)
    	for(j=0;j<N;j++)
           	option.p[i][j]=cholRho[i][j];
    // Timer init
    CudaCheck( hipEventCreate( &d_start ));
    CudaCheck( hipEventCreate( &d_stop ));
    /* CPU Monte Carlo */
    printf("\nMonte Carlo execution on CPU:\nN^ simulations: %d\n\n",SIMS);
    //h_start = clock();
    CudaCheck( hipEventRecord( d_start, 0 ));
    CPU_sim=host_basketOpt(&option, SIMS);
    //h_stop = clock();
    //CPU_timeSpent = ((float)(h_stop - h_start)) / CLOCKS_PER_SEC;
    CudaCheck( hipEventRecord( d_stop, 0));
    CudaCheck( hipEventSynchronize( d_stop ));
    CudaCheck( hipEventElapsedTime( &CPU_timeSpent, d_start, d_stop ));
    CPU_timeSpent /= CLOCKS_PER_SEC;
    h_price = CPU_sim.Expected;

    // GPU Monte Carlo
    printf("\nMonte Carlo execution on GPU:\nN^ simulations: %d\n",SIMS);
    for(i=0; i<THREADS; i++){
    	CudaCheck( hipEventRecord( d_start, 0 ));
       	GPU_sim[i] = dev_basketOpt(&option, numBlocks, numThreads[i])
        CudaCheck( hipEventRecord( d_stop, 0));
        CudaCheck( hipEventSynchronize( d_stop ));
        CudaCheck( hipEventElapsedTime( &GPU_timeSpent[i], d_start, d_stop ));
        GPU_timeSpent[i] /= 1000;
        difference[i] = abs(GPU_sim[i].Expected - h_price);
        speedup[i] = abs(CPU_timeSpent / GPU_timeSpent[i]);
    }
    // Comparing time spent with the two methods
    printf( "\n-\tResults:\t-\n");
    printf("Simulated price for the option with CPU: € %f with I.C. %f\n", h_price, CPU_sim.Confidence);
    printf("Total execution time CPU: %f s with device function\n\n", CPU_timeSpent);
    printf("Simulated price for the option with GPU:\n");
    printf("  : NumThreads : Price : Confidence Interval : Difference from CPU price :  Time : Speedup :");
    printf("\n");
    for(i=0; i<THREADS; i++){
        	printf(": \t %d ",numThreads[i]);
        	printf(" \t %f ",GPU_sim[i].Expected);
        	printf(" \t %f  ",GPU_sim[i].Confidence);
        	printf(" \t %f \t",difference[i]);
        	printf(" \t %f ",GPU_timeSpent[i]);
        	printf(" \t %.2f \t",speedup[i]);
        	printf(":\n");
    }
    CudaCheck( hipEventDestroy( d_start ));
    CudaCheck( hipEventDestroy( d_stop ));
    return 0;
}

void Chol( double c[N][N], double a[N][N] ){
    int i,j,k;
    double v[N];
    for( i=0; i<N; i++){
        for( j=0; j<N; j++ ){
            if( j>=i ){
                //Triangolare inferiore
            	//v[j]=c[j][i]
            	v[j]=c[j][i];
                for(k=0; k<i; k++)    //Scorre tutta
                    //v[j] = v[j] - a[i][k] * a[j][k]
                    v[j] = v[j] - a[i][k] * a[j][k];
                //a[j][i] = v[j] / sqrt( v[i] )
                if(v[i]>0)
                	a[j][i] = v[j] / sqrt( v[i] );
                else
                	a[j][i] = 0.0f;
            }
            else
                //Triangolare superiore a[j][i]
            	a[j][i] = 0.0f;
        }
    }
}
